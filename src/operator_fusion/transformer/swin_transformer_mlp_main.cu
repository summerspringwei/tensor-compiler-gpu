#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>

#include "../../utils.h"
#include "../../cuda_utils.h"

#include "swin_transformer_mlp.h"
#include "swin_transformer_mlp_tvm.h"

void init_values(half* input, half* weight, half* short_cut){
  hf_init_values<half>(input, {524288}, 1, 0);
  hf_init_values<half>(weight, {1048576}, 1, 0);
  hf_init_values<half>(short_cut, {131072}, 1, 0);
}

#define FUNC_TVM default_function_kernel0<<<dim3(8, 8, 1), dim3(32, 1, 4)>>>(d_input, d_weight, d_output, d_short_cut);
#define FUNC_TVM_V2 fc2_16_16_2048_512_tvm_v2<<<dim3(8, 8, 1), dim3(32, 1, 4)>>>(d_input, d_weight, d_output, d_short_cut);
#define FUNC_TVM_V3 hipLaunchCooperativeKernel((void*)fc2_16_16_2048_512_tvm_v3, dim3(8, 8, 1), dim3(32, 1, 4), encoder_kernelArgs, 56*1024);
// #define FUNC fc2_16_16_2048_512_v2<<<dim3(16, 16, 1), dim3(32, 4, 1)>>>(d_input, d_weight, d_output, (half*)nullptr);
// #define FUNC fc2_16_16_2048_512_v3<<<dim3(8, 16, 1), dim3(32, 4, 1)>>>(d_input, d_weight, d_output, (half*)nullptr);

int main(){
	const int input_size=524288;	half *input = new half[input_size];
	const int weight_size=1048576;	half *weight = new half[weight_size];
	const int output_size=131072;	half *output = new half[output_size];
  const int short_cut_size=131072;	half *short_cut = new half[short_cut_size];

  init_values(input, weight, short_cut);

	hipError_t err = hipSuccess;
	half *d_input=NULL;
	half *d_weight=NULL;
	half *d_output=NULL;
  half *d_short_cut=NULL;
	err=hipMalloc((void **)&d_input, sizeof(half)*input_size);
	err=hipMalloc((void **)&d_weight, sizeof(half)*weight_size);
	err=hipMalloc((void **)&d_output, sizeof(half)*output_size);
  err=hipMalloc((void **)&d_short_cut, sizeof(half)*short_cut_size);

	hipMemcpy(d_input, input, sizeof(half)*input_size, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight, sizeof(half)*weight_size, hipMemcpyHostToDevice);
  hipMemcpy(d_short_cut, short_cut, sizeof(half)*short_cut_size, hipMemcpyHostToDevice);
	
  // FUNC_TVM
	hipDeviceSynchronize();
	// hipMemcpy(output, d_output, sizeof(half)*output_size, hipMemcpyDeviceToHost);
  
  // for(int i=0; i<10; ++i){
  //   printf("(%d, %d): %f\n", i/512, i%512, __half2float(output[i]));
  // }
  // for(int i=0; i<output_size; ++i){
	// 	if(__half2float(output[i]) != (float)(2048.0)){
	// 		printf("(%d, %d): %f\n", i/512, i%512, __half2float(output[i]));
	// 	}
  // }printf("\n");
  
  int dev = 0;
  int supportsCoopLaunch = 0;
  hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);

  if(supportsCoopLaunch){
      printf("Device support CoopLaunch\n");
  }
  int numThreads = 64*4, numBlocksPerSm=0; \
  hipDeviceProp_t deviceProp; \
  hipGetDeviceProperties(&deviceProp, dev); \
  hipFuncSetAttribute((void*)fc2_16_16_2048_512_tvm_v3, hipFuncAttributeMaxDynamicSharedMemorySize, 56*1024);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void*)fc2_16_16_2048_512_tvm_v3, numThreads, 0); \
  printf("OccupancyMaxActiveBlocksPerMultiprocessor: %d, multiProcessorCount: %d\n", numBlocksPerSm, deviceProp.multiProcessorCount);\
  
  void *encoder_kernelArgs[] = {(void *)&(d_input), (void *)&(d_weight), (void *)&(d_output), (void *)&(d_short_cut)};
  
  FUNC_TVM_V3
	hipDeviceSynchronize();
  hipMemcpy(output, d_output, sizeof(half)*output_size, hipMemcpyDeviceToHost);
  for(int i=0; i<output_size; ++i){
    // if(i<1024){
    //   printf("%f ", __half2float(output[i]));
    // }
		if( __half2float(output[i]) != 2048){
			printf("(%d, %d): %f\n", i/512, i%512, __half2float(output[i]));
		}
  }printf("\n");

	hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  const int round_cout = 10, loop = 10000;
  float ms = 0, sum = 0;
  // 1. For original pointwise conv
  for(int round =0; round<round_cout; ++round){
    ms = 0, sum = 0;
    for(int i=0; i<loop; ++i){
      checkCuda( hipEventRecord(startEvent,0) );
      // FUNC_TVM
      checkCuda( hipEventRecord(stopEvent,0) );
      checkCuda( hipEventSynchronize(stopEvent) );
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      sum += ms;
    }printf("Before fuse avg time %f\n", sum / loop);
  }

  for(int round =0; round<round_cout; ++round){
    ms = 0, sum = 0;
    for(int i=0; i<loop; ++i){
      checkCuda( hipEventRecord(startEvent,0) );
      FUNC_TVM_V3
      checkCuda( hipEventRecord(stopEvent,0) );
      checkCuda( hipEventSynchronize(stopEvent) );
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      sum += ms;
    }printf("After fuse avg time %f\n", sum / loop);
  }
  
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  delete[] input;
	delete[] weight;
	delete[] output;
  delete[] short_cut;
	hipFree(d_input);
	hipFree(d_weight);
	hipFree(d_output);
  hipFree(d_short_cut);
	return 0;
}