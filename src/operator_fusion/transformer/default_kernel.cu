int main(){
	half *input = new half[524288];
	half *weight = new half[524288];
	half *output = new half[1048576];
	half *intermedia_output = new half[1048576];
	half *ori_output = new half[1048576];

	hipError_t err = hipSuccess;
	half *d_input=NULL;
	half *d_weight=NULL;
	half *d_output=NULL;
	half *d_intermedia_output=NULL;
	half *d_ori_output=NULL;
	err=hipMalloc((void **)&d_input, sizeof(half)*524288);
	err=hipMalloc((void **)&d_weight, sizeof(half)*524288);
	err=hipMalloc((void **)&d_output, sizeof(half)*1048576);
	err=hipMalloc((void **)&d_intermedia_output, sizeof(half)*1048576);
	err=hipMalloc((void **)&d_ori_output, sizeof(half)*1048576);

	hipMemcpy(d_input, input, sizeof(half)*524288, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight, sizeof(half)*524288, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(half)*1048576, hipMemcpyDeviceToHost);
	hipMemcpy(intermedia_output, d_intermedia_output, sizeof(half)*1048576, hipMemcpyDeviceToHost);
	hipMemcpy(ori_output, d_ori_output, sizeof(half)*1048576, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  	delete[] input;
	delete[] weight;
	delete[] output;
	delete[] intermedia_output;
	delete[] ori_output;
	hipFree(d_input);
	hipFree(d_weight);
	hipFree(d_output);
	hipFree(d_intermedia_output);
	hipFree(d_ori_output);
	return 0;
}