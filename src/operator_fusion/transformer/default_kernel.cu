int main(){
	const int input_size=524288;	half *input = new half[input_size];
	const int weight_size=1048576;	half *weight = new half[weight_size];
	const int output_size=4194304;	half *output = new half[output_size];

	hipError_t err = hipSuccess;
	half *d_input=NULL;
	half *d_weight=NULL;
	half *d_output=NULL;
	err=hipMalloc((void **)&d_input, sizeof(half)*input_size);
	err=hipMalloc((void **)&d_weight, sizeof(half)*weight_size);
	err=hipMalloc((void **)&d_output, sizeof(half)*output_size);

	hipMemcpy(d_input, input, sizeof(half)*input_size, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight, sizeof(half)*weight_size, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(half)*output_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  	delete[] input;
	delete[] weight;
	delete[] output;
	hipFree(d_input);
	hipFree(d_weight);
	hipFree(d_output);
	return 0;
}