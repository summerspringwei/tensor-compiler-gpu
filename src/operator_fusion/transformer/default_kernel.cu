
#include <hip/hip_runtime.h>
int main(){
	const int input_size=1024;	float *input = new float[input_size];
	const int weight_size=1024;	float *weight = new float[weight_size];
	const int output_size=256;	float *output = new float[output_size];

	hipError_t err = hipSuccess;
	float *d_input=NULL;
	float *d_weight=NULL;
	float *d_output=NULL;
	err=hipMalloc((void **)&d_input, sizeof(float)*input_size);
	err=hipMalloc((void **)&d_weight, sizeof(float)*weight_size);
	err=hipMalloc((void **)&d_output, sizeof(float)*output_size);

	hipMemcpy(d_input, input, sizeof(float)*input_size, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight, sizeof(float)*weight_size, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(float)*output_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  	delete[] input;
	delete[] weight;
	delete[] output;
	hipFree(d_input);
	hipFree(d_weight);
	hipFree(d_output);
	return 0;
}