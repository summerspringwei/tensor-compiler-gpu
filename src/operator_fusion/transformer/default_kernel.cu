int main(){
	float *input = new float[1024];
	float *weight = new float[2048];
	float *output = new float[1024];

	hipError_t err = hipSuccess;
	float *input=NULL;
	float *weight=NULL;
	float *output=NULL;
	err=hipMalloc((void **)&d_input, sizeof(float)*1024);
	err=hipMalloc((void **)&d_weight, sizeof(float)*2048);
	err=hipMalloc((void **)&d_output, sizeof(float)*1024);

	hipMemcpy(d_input, input, sizeof(float)*1024, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight, sizeof(float)*2048, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipMemcpy(d_output, output, sizeof(float)*1024, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	delete[] input;
	delete[] weight;
	delete[] output;
	hipFree(d_input);
	hipFree(d_weight);
	hipFree(d_output);
	return 0;
}