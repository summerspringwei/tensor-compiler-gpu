#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <iostream>
#include <chrono>
#include <vector>

#include <hipDNN.h>
#include <hip/hip_fp16.h>
#include <cuda_fp16.hpp>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "../../cuda_utils.h"

#define CUDNN_CALL(f) { \
  hipdnnStatus_t err = (f); \
  if (err != HIPDNN_STATUS_SUCCESS) { \
    std::cout \
        << "    Error occurred: " << err << " at " << __LINE__ << std::endl; \
    std::exit(1); \
  } \
}

int64_t get_shape_size(std::vector<int64_t> shape){
  int64_t acc = 1;
  for(auto s: shape){
    acc *= s;
  }
  return acc;
}

// Old implementation for swin-transformer with shape (64, 4, 49, 49)
void bench_softmax(std::vector<int64_t> shape){
  const int shape_size = get_shape_size(shape);
  half *input = new half[shape_size];
  half *output = new half[shape_size];

  hipError_t err = hipSuccess;
  half *d_input=NULL;
  half *d_output=NULL;
  half *d_alpha=NULL;
  half *d_beta=NULL;
  err=hipMalloc((void **)&d_input, sizeof(half)*shape_size);
  err=hipMalloc((void **)&d_output, sizeof(half)*shape_size);

  hipMemcpy(d_input, input, sizeof(half)*shape_size, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  hipMemcpy(output, d_output, sizeof(half)*shape_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  hipdnnHandle_t handle;
  CUDNN_CALL(hipdnnCreate(&handle));
  hipdnnTensorDescriptor_t x_desc, y_desc;
  hipdnnCreateTensorDescriptor(&x_desc);
  hipdnnCreateTensorDescriptor(&y_desc);
  hipdnnSetTensor4dDescriptor(
        x_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF,
        shape[0], shape[1], shape[2], shape[3]);
  hipdnnSetTensor4dDescriptor(
        y_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_HALF,
        shape[0], shape[1], shape[2], shape[3]);
  int loop=10000;
  half alpha=1.0, beta=0.0;
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  
  for(int a=0; a<3; ++a){
    float sum = 0, ms=0;
    auto t1 = std::chrono::steady_clock::now();
    for(int i=0; i<loop; ++i){

      // hipEventRecord(startEvent,0);
      auto result = hipdnnSoftmaxForward(handle, 
        hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_ACCURATE, 
        hipdnnSoftmaxMode_t::HIPDNN_SOFTMAX_MODE_CHANNEL, 
        &alpha, x_desc, d_input, &beta, y_desc, d_output);
      // checkCuda( hipEventRecord(stopEvent,0) );
      // checkCuda( hipEventSynchronize(stopEvent) );
      // checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      // sum += ms;
    }
    auto t2 = std::chrono::steady_clock::now();
    double latency = std::chrono::duration<double, std::micro>(t2-t1).count();
    printf("chrono: %f\n", latency / loop);
    // printf("%f\n", sum/loop);
  }

  hipdnnDestroy(handle);

  delete[] input;
  delete[] output;
  hipFree(d_input);
  hipFree(d_output);
}


void bench_query_key(){
  hipblasStatus_t hipblasHgemm(hipblasHandle_t handle,
                           hipblasOperation_t transa, hipblasOperation_t transb,
                           int m, int n, int k,
                           const __half *alpha,
                           const __half *A, int lda,
                           const __half *B, int ldb,
                           const __half *beta,
                           __half *C, int ldc);
  const int input_size=64*4*64*32;     half *input = new half[input_size];
  const int weight_size=64*4*64*32;    half *weight = new half[weight_size];
  const int output_size=64*4*64*64;    half *output = new half[output_size];

  hipError_t err = hipSuccess;
  half *d_input=NULL;
  half *d_weight=NULL;
  half *d_output=NULL;
  err=hipMalloc((void **)&d_input, sizeof(half)*input_size);
  err=hipMalloc((void **)&d_weight, sizeof(half)*weight_size);
  err=hipMalloc((void **)&d_output, sizeof(half)*output_size);

  hipMemcpy(d_input, input, sizeof(half)*input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_weight, weight, sizeof(half)*weight_size, hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  hipMemcpy(output, d_output, sizeof(half)*output_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  half alpha = 1.0, beta=0.0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  // int m = 49, n = 49, k = 32;
  int m = 64, n = 64, k = 32;
  int batch_size = 64 * 4;
  const void * Aarray[batch_size];
  const void * Barray[batch_size];
  void * Carray[batch_size];
  for(int i=0; i<batch_size; ++i){
    Aarray[i] = d_input + i * m * k;
    Barray[i] = d_weight + i * n * k;
    Carray[i] = d_output + i * m * n;
  }
  
  int loop=10000;
  for(int a=0; a<3; ++a){
    auto t1 = std::chrono::steady_clock::now();
    for(int i=0; i<loop; ++i){
      // hipblasHgemmBatched(handle, 
      //   hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T, 
      //   m, n, k,
      //   &alpha,
      //   Aarray, m,
      //   Barray, n,
      //   &beta,
      //   Carray, m,
      //   batch_size
      // );
      hipblasGemmBatchedEx(handle, 
        hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T, 
        m, n, k,
        (const void *)&alpha,
        Aarray, hipDataType::HIP_R_16F, m,
        Barray, hipDataType::HIP_R_16F, n,
        &beta,
        Carray, hipDataType::HIP_R_16F, m,
        batch_size,
        hipblasComputeType_t::HIPBLAS_COMPUTE_16F,
        hipblasGemmAlgo_t::CUBLAS_GEMM_DEFAULT_TENSOR_OP
      );
    }
    auto t2 = std::chrono::steady_clock::now();
    double latency = std::chrono::duration<double, std::micro>(t2-t1).count();
    printf("avg chrono: %f\n", latency / loop);
  }
  delete[] input;
  delete[] weight;
  delete[] output;
  hipFree(d_input);
  hipFree(d_weight);
  hipFree(d_output);
}

template<typename T>
void bench_matmul(int m, int n, int k){

  int input_size = m * k;
  int weight_size = n * k;
  int output_size = m * n;

  half *input = new half[input_size];
  half *weight = new half[weight_size];
  half *output = new half[output_size];

  hipError_t err = hipSuccess;
  void *d_input=NULL;
  void *d_weight=NULL;
  void *d_output=NULL;
  err=hipMalloc((void **)&d_input, sizeof(T)*input_size);
  err=hipMalloc((void **)&d_weight, sizeof(T)*weight_size);
  err=hipMalloc((void **)&d_output, sizeof(T)*output_size);

  hipMemcpy((void*)d_input, input, sizeof(T)*input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_weight, weight, sizeof(T)*weight_size, hipMemcpyHostToDevice);

  T alpha = 1.0, beta=0.0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipDataType cuda_dtype=hipDataType::HIP_R_32F;
  if(std::is_same<T, half>::value){
    cuda_dtype = hipDataType::HIP_R_16F;
  }else if(std::is_same<T, float>::value){
    cuda_dtype = hipDataType::HIP_R_32F;
  }
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  int loop=1;
  double min_latency = 1e9;
  for(int a=0; a<1; ++a){
    auto t1 = std::chrono::steady_clock::now();
    float sum = 0;
    for(int i=0; i<loop; ++i){
      float ms = 0;
      hipEventRecord(startEvent,0);
      auto cublasStat = hipblasGemmEx(handle, 
        hipblasOperation_t::HIPBLAS_OP_N, hipblasOperation_t::HIPBLAS_OP_T, 
        m, n, k, 
        (const void *)&alpha, 
        d_input, cuda_dtype, m, 
        d_weight, cuda_dtype, n, 
        (const void *)&beta, 
        d_output, cuda_dtype, m, 
        cuda_dtype, 
        hipblasGemmAlgo_t::CUBLAS_GEMM_DEFAULT_TENSOR_OP);

        checkCuda( hipEventRecord(stopEvent,0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
        sum += ms;
    }
    printf("sum: %f\n", sum/loop);
    auto t2 = std::chrono::steady_clock::now();
    double latency = std::chrono::duration<double, std::micro>(t2-t1).count() / loop;
    if(latency < min_latency){
      min_latency = latency;
    }
  }
  printf("avg chrono: %f\n", min_latency);
  hipDeviceSynchronize();
  hipMemcpy(output, d_output, sizeof(half)*output_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  delete[] input;
  delete[] weight;
  delete[] output;
  hipFree(d_input);
  hipFree(d_weight);
  hipFree(d_output);
}


void bench_qvk_matmul(int batch_size, int height, int width, int channel){
  bench_matmul<half>(batch_size * height * width, 3*channel, channel);
}

void bench_unfused_qvk_matmul(int batch_size, int height, int width, int channel){
  bench_matmul<half>(batch_size * height * width, channel, channel);
}


void bench_FFN_fc1(int batch_size, int height, int width, int channel){
  bench_matmul<half>(batch_size * height * width, 4*channel, channel);
}

void bench_FFN_fc2(int batch_size, int height, int width, int channel){
  bench_matmul<half>(batch_size * height * width, channel, 4*channel);
}




int main(){
  // bench_softmax({1, 1, 128, 768});
  // bench_query_key();
  // bench_qvk_matmul(1, 64, 64, 128);
  // bench_qvk_matmul(1, 384, 768, 768);
  bench_unfused_qvk_matmul(1, 384, 768, 768);
  // bench_qvk_matmul(1, 32, 32, 256);
  // bench_qvk_matmul(1, 16, 16, 512);
  // bench_qvk_matmul(1, 8, 8, 1024);
  // bench_FFN_fc1(1, 64, 64, 1280);
  // bench_FFN_fc1(1, 32, 32, 256);
  // bench_FFN_fc1(1, 16, 16, 512);
  // bench_FFN_fc1(1, 8, 8, 1024);
  // For nerf model
  // bench_matmul<half>(108*4*16, 256, 256);
  // bench_matmul<half>(65536, 256, 256);
  // bench_FFN_fc2(1, 64, 64, 128);
  // bench_FFN_fc2(1, 32, 32, 256);
  // bench_FFN_fc2(1, 16, 16, 512);
  // bench_FFN_fc2(1, 8, 8, 1024);
  return 0;
}


