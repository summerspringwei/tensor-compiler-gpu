#include "hip/hip_runtime.h"
// attr [IterVar(blockIdx.x: int32, (nullptr), "ThreadIndex", "blockIdx.x")] "thread_extent" = 24;
// attr [IterVar(blockIdx.y: int32, (nullptr), "ThreadIndex", "blockIdx.y")] "thread_extent" = 1 
// attr [IterVar(blockIdx.z: int32, (nullptr), "ThreadIndex", "blockIdx.z")] "thread_extent" = 12;
// attr [IterVar(threadIdx.z: int32, (nullptr), "ThreadIndex", "threadIdx.z")] "thread_extent" = 6;
// attr [IterVar(threadIdx.y: int32, (nullptr), "ThreadIndex", "threadIdx.y")] "thread_extent" = 1;
// attr [IterVar(threadIdx.x: int32, (nullptr), "ThreadIndex", "threadIdx.x")] "thread_extent" = 32;
// dim3(24, 1, 12), dim3(32, 1, 6)

#include "transformer_query_key_softmax.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include "../../utils.h"



void init_input_weight(half* input, half* weight, half* output){
  hf_init_values<half>(input, {12, 384, 64}, __float2half(.1));
  hf_init_values<half>(weight, {12, 384, 64}, __float2half(.1));
  hf_init_values<half>(output, {12, 384, 64}, __float2half(0.0));
}


#define FUNC1 query_key_matmul<<<dim3(6, 12, 12), dim3(32, 2, 1)>>>(d_input, d_weight, d_intermedia_output);
#define FUNC2 fused_mul_softmax<<<dim3(576, 1, 1), dim3(256, 1, 1)>>>(d_intermedia_output, d_ori_output);

#define FUSED_FUNCV1 qeury_key_matmul_softmax<<<dim3(24, 12, 1), dim3(32, 1, 6)>>>(d_input, d_weight, d_output);
// #define FUSED_FUNC fused_qeury_key_matmul_softmax<<<dim3(24, 12, 1), dim3(32, 1, 6)>>>(d_input, d_weight, d_output);
#define FUSED_FUNC fused_qeury_key_matmul_softmax_v2<<<dim3(24, 12, 1), dim3(32, 1, 6)>>>(d_input, d_weight, d_output);


int main(){
	half *input = new half[296448];
	half *weight = new half[296448];
	half *output = new half[1769472];
	half *intermedia_output = new half[1769472];
	half *ori_output = new half[1769472];

	hipError_t err = hipSuccess;
	half *d_input=NULL;
	half *d_weight=NULL;
	half *d_output=NULL;
	half *d_intermedia_output=NULL;
	half *d_ori_output=NULL;
	err=hipMalloc((void **)&d_input, sizeof(half)*296448);
	err=hipMalloc((void **)&d_weight, sizeof(half)*296448);
	err=hipMalloc((void **)&d_output, sizeof(half)*1769472);
	err=hipMalloc((void **)&d_intermedia_output, sizeof(half)*1769472);
	err=hipMalloc((void **)&d_ori_output, sizeof(half)*1769472);

	hipMemcpy(d_input, input, sizeof(half)*296448, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, weight, sizeof(half)*296448, hipMemcpyHostToDevice);
  // FUNC1
  FUNC2
  FUSED_FUNC
	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(half)*1769472, hipMemcpyDeviceToHost);
	hipMemcpy(intermedia_output, d_intermedia_output, sizeof(half)*1769472, hipMemcpyDeviceToHost);
	hipMemcpy(ori_output, d_ori_output, sizeof(half)*1769472, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
  // Benchmark
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  const int round_cout = 2, loop = 10000;
  float ms = 0, sum = 0;
  // 1. For original pointwise conv
  for(int round =0; round<round_cout; ++round){
    ms = 0, sum = 0;
    for(int i=0; i<loop; ++i){
      checkCuda( hipEventRecord(startEvent,0) );
      // FUNC1
      FUNC2
      checkCuda( hipEventRecord(stopEvent,0) );
      checkCuda( hipEventSynchronize(stopEvent) );
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      sum += ms;
    }printf("Before fuse avg time %f\n", sum / loop);
  }
  
  for(int round =0; round<round_cout; ++round){
    ms = 0, sum = 0;
    for(int i=0; i<loop; ++i){
      checkCuda( hipEventRecord(startEvent,0) );
      FUSED_FUNC
      checkCuda( hipEventRecord(stopEvent,0) );
      checkCuda( hipEventSynchronize(stopEvent) );
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      sum += ms;
    }printf("After fuse avg time %f\n", sum / loop);
  }
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  for(int i=0;i<10;++i){
    printf("%.6f ", __half2float(output[i]));
  }printf("\n");
  for(int i=0;i<10;++i){
    printf("%.6f ", __half2float(ori_output[i]));
  }printf("\n");

  delete[] input;
	delete[] weight;
	delete[] output;
	delete[] intermedia_output;
	delete[] ori_output;
	hipFree(d_input);
	hipFree(d_weight);
	hipFree(d_output);
	hipFree(d_intermedia_output);
	hipFree(d_ori_output);
	return 0;
}