#include "hip/hip_runtime.h"
#include <mma.h>

#include <iostream>
#include <vector>
#include <algorithm>

#include "torch/all.h"

#include "../../cuda_utils.h"
#include "../../utils.h"
#include "../torch_utils.h"

#include "gpt2-large.h"
#include "kernels/gemm.cu"
#include "kernels/fused_gpt2_attn.cu"


template <int64_t batch_size, int64_t num_heads, int64_t max_seq_length,
          int64_t hidden_size, int64_t d_intermedia>

class Attn {
    public:
    Attn(std::string folder_path, torch::Tensor input_tensor){
        this->folder_path = folder_path;
        this->input_tensor = torch::permute(input_tensor, {1, 0, 2}).contiguous();
        load_weight();
        init_intermedia_tensor();
        init_tensor_pointers();
    }
    ~Attn(){}

    void load_weight() {
        this->qkv_weight =
        torch_load_tensor(folder_path + "gpt2-torch-data/c_attn.pt")
            .to(torch::kCUDA)
            .to(torch::kHalf);
        this->attn_fc_weight =
            torch_load_tensor(folder_path + "gpt2-torch-data/attn_c_proj.pt")
                .to(torch::kCUDA)
                .to(torch::kHalf);
        this->permuted_qkv_weight = torch::reshape(torch::permute(qkv_weight, {1, 0}), {3, d_model, d_model}).contiguous();// 3840, 1280 -> 3 * 1280, 1280
    }

    void init_intermedia_tensor() {
        this->t_attn_mask = torch::zeros({batch_size*num_heads, max_seq_length, max_seq_length}, options_fp16);
        // this->output_qkv = torch::zeros({batch_size*3, num_heads, max_seq_length, hidden_size}, options_fp16);
        this->output_qkv = torch::zeros({3*max_seq_length, d_model}, options_fp16);
        this->qkv_bias = torch::zeros({3, d_model}, options_fp16);
        this->query_key_output = torch::zeros({batch_size*num_heads, max_seq_length, max_seq_length}, options_fp16);
        this->query_key_softmax_sum = torch::zeros({batch_size*num_heads, max_seq_length}, options_fp32);
        this->attn_value_output = torch::zeros({batch_size*max_seq_length, d_model}, options_fp16);
        this->attn_fc_output = torch::zeros({batch_size*max_seq_length, d_model}, options_fp16);
        float v_d_model[] = {d_model,};
        this->t_d_model = torch::from_blob(v_d_model, {1,}).to(torch::kCUDA);
        this->layer_norm_sum = torch::zeros({batch_size*max_seq_length,}, options_fp32);
        this->layer_norm_variance = torch::zeros({batch_size*max_seq_length,}, options_fp32);
    }

    void init_tensor_pointers() {
        this->ptr_input_tensor = (at::Half*)this->input_tensor.data_ptr<at::Half>();
        // this->ptr_qkv_weight = (at::Half*)this->qkv_weight.data_ptr<at::Half>();
        this->ptr_qkv_weight = (at::Half*)this->permuted_qkv_weight.data_ptr<at::Half>();
        this->ptr_qkv_bias = (at::Half*)this->qkv_bias.data_ptr<at::Half>();
        this->ptr_output_qkv = (at::Half*)this->output_qkv.data_ptr<at::Half>();
        // this->ptr_query = this->ptr_output_qkv + (max_seq_length * d_model);
        // this->ptr_key = this->ptr_query + (max_seq_length * d_model);
        // this->ptr_value = this->ptr_key + (max_seq_length * d_model);
        
        this->ptr_query_key_output = this->query_key_output.data_ptr<at::Half>();
        this->ptr_t_attn_mask = this->t_attn_mask.data_ptr<at::Half>();
        this->ptr_query_key_softmax_sum = this->query_key_softmax_sum.data_ptr<float>();
        this->ptr_attn_value_output = this->attn_value_output.data_ptr<at::Half>();
        this->ptr_attn_fc_weight = this->attn_fc_weight.data_ptr<at::Half>();
        this->ptr_attn_fc_output = this->attn_fc_output.data_ptr<at::Half>();
        this->ptr_layer_norm_sum = this->layer_norm_sum.data_ptr<float>();
        this->ptr_layer_norm_variance = this->layer_norm_variance.data_ptr<float>();
    }

    void torch_forward() {
        auto batched_src = torch::reshape(input_tensor.repeat({3, 1, 1}), {3, max_seq_length, d_model});
        // auto torch_permuted_qkv_weight = torch::permute(qkv_weight.reshape({d_model, 3, d_model}), {1, 0, 2});
        auto torch_permuted_qkv_weight = torch::reshape(torch::permute(qkv_weight, {1, 0}), {3, d_model, d_model});// (3*1280r, 1280)
        printf("torch_permuted_qkv_weight shape: %s\n ", get_torch_tensor_shape_str(torch_permuted_qkv_weight).c_str());
        this->bmm_output = torch::bmm(batched_src, torch_permuted_qkv_weight);// (3, seq_length, d_model)
        auto t_output_qkv = torch::permute(torch::reshape(bmm_output,
                {3, max_seq_length, num_heads, hidden_size}), {0, 2, 1, 3});// (3, num_heads, seq_length, hidden_size)
        t_qkv = torch::split(t_output_qkv, 1, 0);
        t_query = torch::reshape(t_qkv[0], {batch_size*num_heads, max_seq_length, hidden_size}).contiguous();
        t_key = torch::reshape(t_qkv[1], {batch_size*num_heads, max_seq_length, hidden_size}).contiguous();
        t_value = torch::reshape(t_qkv[2], {batch_size*num_heads, max_seq_length, hidden_size}).contiguous();
        // (20, 384, 64) * （20， 384， 64) -> (20, 384, 384)
        t_query_key_output = t_query.bmm(torch::permute(t_key, {0, 2, 1}));
        t_query_key_softmax = torch::softmax(
            (t_query_key_output / torch::sqrt(t_d_model)) + 
            t_attn_mask, -1, torch::kHalf);
        t_attn_value_output = torch::bmm(t_query_key_softmax, t_value); 
        t_attn_value_output_permuted = torch::reshape(
            torch::permute(t_attn_value_output, {1, 0, 2}), {max_seq_length, d_model});
        t_attn_fc_output = torch::matmul(t_attn_value_output_permuted, attn_fc_weight);
        t_attn_fc_short_cut_add = t_attn_fc_output + input_tensor;
        auto t_layer_norm_sum = torch::sum(t_attn_fc_short_cut_add, -1);
        auto t_layer_norm_mean = t_layer_norm_sum / d_model;
        auto expand_t_layer_norm_mean = t_layer_norm_mean.reshape({batch_size, max_seq_length, 1}).repeat({1, 1, d_model});
        auto t_layer_norm_variance = torch::sum(
            torch::pow(t_attn_fc_short_cut_add - expand_t_layer_norm_mean, 2), -1) / d_model;
        printf("\n t_layer_norm_sum\n ");
        torch::print(t_layer_norm_sum);
        printf("\n t_layer_norm_variance\n ");
        torch::print(t_layer_norm_variance);
        t_attn_fc_layernorm_output = torch::layer_norm(t_attn_fc_short_cut_add, {d_model,});
    }

    void qkv(){
        printf("permuted_qkv_weight shape: %s\n ", get_torch_tensor_shape_str(this->permuted_qkv_weight).c_str());
        // auto weight = torch::ones({3, d_model, d_model}, options_fp16) / 16;
        // at::Half* ptr_weight = weight.data_ptr<at::Half>();
        // auto input = torch::ones({3, max_seq_length, d_model}, options_fp16);
        // at::Half* ptr_input = input.data_ptr<at::Half>();
        void* fused_attn_kernel_args[] = {(void *)&(ptr_qkv_weight), (void *)&(ptr_input_tensor), 
            (void *)&(ptr_qkv_bias), (void *)&(ptr_output_qkv)
        };
        // void* fused_attn_kernel_args[] = {(void *)&(ptr_weight), (void *)&(ptr_input), 
        //     (void *)&(ptr_qkv_bias), (void *)&(ptr_output_qkv)
        // };
        checkCuda(hipFuncSetAttribute((void*)gemm_add_qkv_bias, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnQKVParams::kSharedMemory), __LINE__);
        checkCuda(hipLaunchKernel((void*)gemm_add_qkv_bias,
            dim3(AttnQKVParams::kGridBlocks, 1, 1), dim3(AttnQKVParams::kBlockThreads, 1, 1), 
            fused_attn_kernel_args, AttnQKVParams::kSharedMemory), __LINE__);
    }

    void query_key() {
        this->ptr_query = this->t_query.data_ptr<at::Half>();
        this->ptr_key = this->t_key.data_ptr<at::Half>();
        this->ptr_query_key_output = this->query_key_output.data_ptr<at::Half>();
        void* args[] = {
            (void *)&(ptr_key), (void *)&(ptr_query), (void *)&(ptr_query_key_output)
        };
        
        printf("blocks %d, shared memory: %d\n ", AttnQueryKeyParams::kGridBlocks, AttnQueryKeyParams::kSharedMemory);
        checkCuda(hipFuncSetAttribute((void*)gemm_k2, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnQueryKeyParams::kSharedMemory), __LINE__);
        checkCuda(hipLaunchKernel((void*)gemm_k2,
            dim3(AttnQueryKeyParams::kGridBlocks, 1, 1), dim3(AttnQueryKeyParams::kBlockThreads, 1, 1), 
            args, AttnQueryKeyParams::kSharedMemory), __LINE__);
    }


    void query_key_limited_blocks() {
        this->ptr_query = this->t_query.data_ptr<at::Half>();
        this->ptr_key = this->t_key.data_ptr<at::Half>();
        this->ptr_query_key_output = this->query_key_output.data_ptr<at::Half>();
        void* args[] = {
            (void *)&(ptr_key), (void *)&(ptr_query), (void *)&(ptr_query_key_output)
        };
        printf("blocks %d, shared memory: %d\n ", AttnQueryKeyParamsLimitedBlocks::kGridBlocks, AttnQueryKeyParamsLimitedBlocks::kSharedMemory);
        checkCuda(hipFuncSetAttribute((void*)gemm_k2_limited_blocks, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnQueryKeyParamsLimitedBlocks::kSharedMemory), __LINE__);
        checkCuda(hipLaunchKernel((void*)gemm_k2_limited_blocks,
            dim3(AttnQueryKeyParamsLimitedBlocks::kGridBlocks, 1, 1), dim3(AttnQueryKeyParamsLimitedBlocks::kBlockThreads, 1, 1), 
            args, AttnQueryKeyParamsLimitedBlocks::kSharedMemory), __LINE__);
    }

    void query_key_limited_blocks_div_softmax(){
        this->ptr_query = this->t_query.data_ptr<at::Half>();
        this->ptr_key = this->t_key.data_ptr<at::Half>();
        this->ptr_query_key_softmax_sum = this->query_key_softmax_sum.data_ptr<float>();
        this->ptr_query_key_output = this->query_key_output.data_ptr<at::Half>();
        void* args[] = {
            (void *)&(ptr_key), (void *)&(ptr_query), (void*)&(ptr_query_key_softmax_sum), (void *)&(ptr_query_key_output)
        };

        printf("blocks %d, shared memory: %d\n ", AttnQueryKeyParamsLimitedBlocks::kGridBlocks, AttnQueryKeyParamsLimitedBlocks::kSharedMemory);
        checkCuda(hipFuncSetAttribute((void*)gemm_k2_limited_blocks_div_softmax, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnQueryKeyParamsLimitedBlocks::kSharedMemory), __LINE__);
        checkCuda(hipLaunchCooperativeKernel((void*)gemm_k2_limited_blocks_div_softmax,
            dim3(AttnQueryKeyParamsLimitedBlocks::kGridBlocks, 1, 1), dim3(AttnQueryKeyParamsLimitedBlocks::kBlockThreads, 1, 1), 
            args, AttnQueryKeyParamsLimitedBlocks::kSharedMemory), __LINE__);
    }
    
    void attn_value() {
        this->ptr_value = this->t_value.data_ptr<at::Half>();
        this->ptr_query_key_output = this->t_query_key_softmax.data_ptr<at::Half>();
        void* args[] = {
            (void *)&(ptr_value), (void *)&(ptr_query_key_output), 
            (void *)&(ptr_attn_value_output), 
        };
        printf("blocks %d, shared memory: %d\n ", AttnValueParams::kGridBlocks, AttnValueParams::kSharedMemory);
        checkCuda(hipFuncSetAttribute((void*)gemm_reshape, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnValueParams::kSharedMemory), __LINE__);
        checkCuda(hipLaunchKernel((void*)gemm_reshape, 
            dim3(AttnValueParams::kGridBlocks, 1, 1), dim3(AttnValueParams::kBlockThreads, 1, 1), args, AttnValueParams::kSharedMemory));
    }

    void attn_fc() {
        this->ptr_attn_value_output = t_attn_value_output_permuted.data_ptr<at::Half>();
        void* args [] = {
            (void*)&ptr_attn_fc_weight, (void*)&ptr_attn_value_output, (void*)&ptr_attn_fc_output
        };
        printf("blocks %d, shared memory: %d\n ", AttnFcParams::kGridBlocks, AttnFcParams::kSharedMemory);
        void* func_kernel = (void*)gemm_three_stage<AttnFcParams::kGemmK4WarpRowTiles, AttnFcParams::kGemmK4WarpColTiles, 
            kHeadNum * kHeadSize, kSeqLength, kHeadNum * kHeadSize, 1>;
        checkCuda(hipFuncSetAttribute((void*)func_kernel, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnFcParams::kSharedMemory), __LINE__);
        checkCuda(hipLaunchKernel((void*)func_kernel,
            dim3(AttnFcParams::kGridBlocks, 1, 1), dim3(AttnFcParams::kBlockThreads, 1, 1), 
            args, AttnFcParams::kSharedMemory), __LINE__);
    }

    void attn_fc_short_cut_layer_norm() {
        this->ptr_attn_value_output = t_attn_value_output_permuted.data_ptr<at::Half>();
        void* args [] = {
            (void*)&ptr_attn_fc_weight, (void*)&ptr_attn_value_output, (void*)&ptr_input_tensor,
            (void*)&ptr_layer_norm_sum, (void*)&ptr_layer_norm_variance, 
            (void*)&eps, (void*)&gama, (void*)&beta, (void*)&ptr_attn_fc_output,
        };
        printf("blocks %d, shared memory: %d\n ", AttnFcParams::kGridBlocks, AttnFcParams::kSharedMemory);
        checkCuda(hipFuncSetAttribute((void*)attn_fc_layer_norm, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, AttnFcParams::kSharedMemory), __LINE__);
        checkCuda(hipLaunchCooperativeKernel((void*)attn_fc_layer_norm,
            dim3(AttnFcParams::kGridBlocks, 1, 1), dim3(AttnFcParams::kBlockThreads, 1, 1), 
            args, AttnFcParams::kSharedMemory), __LINE__);
    }

    void fused_attn() {
        void* args[] = {
            (void*)&ptr_qkv_weight, 
            (void*)&ptr_input_tensor, 
            (void*)&ptr_qkv_bias, 
            (void*)&ptr_output_qkv,
            (void*)&ptr_key, (void*)&ptr_query, 
            (void*)&ptr_query_key_softmax_sum, 
            (void*)&ptr_query_key_output,
            (void*)&ptr_value,
            (void*)&ptr_attn_value_output,
            (void*)&ptr_attn_fc_weight,
            (void*)&ptr_layer_norm_sum,
            (void*)&ptr_layer_norm_variance,
            (void*)&eps, (void*)&gama, (void*)&beta, (void*)&ptr_attn_fc_output
        };
        std::vector<int> shared_memorys = {
            AttnQKVParams::kSharedMemory, 
            AttnQueryKeyParamsLimitedBlocks::kSharedMemory,
            AttnValueParams::kSharedMemory,
            AttnFcParams::kSharedMemory
        };
        std::vector<int> vec_blocks = {
            AttnQKVParams::kGridBlocks, 
            AttnQueryKeyParamsLimitedBlocks::kGridBlocks,
            AttnValueParams::kGridBlocks,
            AttnFcParams::kGridBlocks
        };
        auto it = std::max_element(std::begin(shared_memorys), std::end(shared_memorys)); // C++11
        const int kSharedMemory = *it;
        it = std::max_element(std::begin(vec_blocks), std::end(vec_blocks)); // C++11
        const int kGridBlocks = *it;
        const int kBlockThreads = 128;
        checkCuda(hipFuncSetAttribute((void*)fused_gpt2_attn, 
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize, kSharedMemory), __LINE__);
        checkCuda(hipLaunchCooperativeKernel((void*)fused_gpt2_attn,
            dim3(kGridBlocks, 1, 1), dim3(kBlockThreads, 1, 1), args, kSharedMemory), __LINE__);
    }

    void souffle_forward() {
        qkv();
        // query_key();
        // query_key_limited_blocks();
        query_key_limited_blocks_div_softmax();
        
        attn_value();
        // attn_fc();
        attn_fc_short_cut_layer_norm();

        fused_attn();
    }

    

    void print() {
        printf("\n input_tensor:\n ");
        torch::print(input_tensor);
        printf("\n qkv_weight:\n ");
        torch::print(qkv_weight);
        printf("\n t_query:\n ");
        // torch::print(t_query);
        torch::print(t_qkv[0]);
        printf("\n t_key:\n ");
        // torch::print(t_key);
        torch::print(t_qkv[1]);
        printf("\n t_value:\n ");
        // torch::print(t_value);
        torch::print(t_qkv[2]);
        printf("\n bmm_output:\n ");
        torch::print(bmm_output);
        printf("\n output_qkv\n ");
        torch::print(output_qkv);
        
        printf("\n t_query_key_output:\n ");
        torch::print(t_query_key_output);
        printf("\n t_query_key_softmax\n ");
        torch::print(t_query_key_softmax);
        printf("\n query_key_output:\n ");
        torch::print(query_key_output);

        printf("\n t_attn_value_output:\n ");
        torch::print(t_attn_value_output_permuted);
        printf("\n attn_value_output:\n ");
        torch::print(attn_value_output);
        printf("\n t_attn_fc_output:\n ");
        torch::print(t_attn_fc_output);
        printf("\n attn_fc_output:\n ");
        torch::print(attn_fc_output);
        // assert(torch::allclose(t_attn_fc_output, attn_fc_output, 1e-1, 1e-1));
        my_compare(t_attn_fc_output, attn_fc_output, 1e-1, 1e-1);
        printf("\n t_attn_fc_short_cut_add:\n ");
        torch::print(t_attn_fc_short_cut_add);
        printf("\n t_attn_fc_layernorm_output\n ");
        torch::print(t_attn_fc_layernorm_output);
        printf("\n layer_norm_sum\n ");
        torch::print(layer_norm_sum);
        printf("\n layer_norm_variance\n ");
        torch::print(layer_norm_variance);
        
    }

public:
    const int64_t d_model = num_heads * hidden_size;
    std::string folder_path;
    // Torch tensors
    torch::Tensor input_tensor;
    std::vector<at::Tensor> t_qkv;
    torch::Tensor permuted_qkv_weight;
    torch::Tensor bmm_output;
    torch::Tensor t_query;
    torch::Tensor t_key;
    torch::Tensor t_value;
    torch::Tensor t_query_key_output;
    torch::Tensor t_attn_mask;
    torch::Tensor t_d_model;
    torch::Tensor t_query_key_softmax;
    torch::Tensor t_attn_value_output;
    torch::Tensor t_attn_value_output_permuted;
    torch::Tensor t_attn_fc_output;
    torch::Tensor t_attn_fc_short_cut_add;
    torch::Tensor t_attn_fc_layernorm_output;

    // Our tensors
    torch::Tensor output_qkv;
    torch::Tensor query_key_output;
    torch::Tensor query_key_softmax_sum;
    torch::Tensor attn_value_output;
    torch::Tensor attn_fc_output;
    // Weights
    torch::Tensor qkv_weight;
    torch::Tensor qkv_bias;
    torch::Tensor attn_fc_weight;
    torch::Tensor layer_norm_sum;
    torch::Tensor layer_norm_variance;
    // Our pointers
    at::Half* ptr_input_tensor;
    at::Half* ptr_qkv_weight;
    at::Half* ptr_qkv_bias;
    at::Half* ptr_output_qkv;
    at::Half* ptr_query;
    at::Half* ptr_key;
    at::Half* ptr_value;
    at::Half* ptr_query_key_output;
    at::Half* ptr_t_attn_mask;
    float* ptr_query_key_softmax_sum;
    at::Half* ptr_attn_value_output;
    at::Half* ptr_attn_fc_weight;
    at::Half* ptr_attn_fc_output;
    float* ptr_layer_norm_sum;
    float* ptr_layer_norm_variance;

    half eps = 0.00001, gama = 1, beta = 0;
};


int main(int argc, char* argv) {
    std::string folder_path =
      "/home/xiachunwei/Projects/tensor-compiler-gpu/src/operator_fusion/gpt-2/";
    torch::Tensor input_tensor = 
      torch_load_tensor(folder_path + "gpt2-torch-data/attn_input_hidden_states.pt")
          .to(torch::kCUDA)
          .to(torch::kHalf);
    Attn<1, 20, 384, 64, 5120> module_attn(
      folder_path, input_tensor);
    
    module_attn.torch_forward();
    module_attn.souffle_forward();
    module_attn.print();
    return 0;
}
