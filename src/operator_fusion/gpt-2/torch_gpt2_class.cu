#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <sstream>
#include <vector>

#include "torch/all.h"

#include "../../cuda_utils.h"
#include "../../utils.h"
#include "../torch_utils.h"

#include "gpt2-large.h"
#include "kernels/gemm.cu"
#include "kernels/fused_feed_forward_pipeline.cu"
#include "kernels/fused_feed_forward_seq.cu"
#include "kernels/layer_norm.cu"

using namespace souffle::gpt2;

template <int64_t batch_size, int64_t num_heads, int64_t max_seq_length,
          int64_t hidden_size, int64_t d_intermedia>
class FeedForward {
 public:
  FeedForward(std::string folder_path, torch::Tensor input_tensor) {
    this->folder_path = folder_path;
    this->input_tensor = input_tensor;
    load_weight();
    init_intermedia_tensor();
    init_tensor_pointers();
  }
  ~FeedForward() {}

  void load_weight() {
    this->feed_forward_fc1_weight =
        torch_load_tensor(folder_path + "gpt2-torch-data/MLP_c_fc.pt")
            .to(torch::kCUDA)
            .to(torch::kHalf);
    this->feed_forward_fc2_weight =
        torch_load_tensor(folder_path + "gpt2-torch-data/MLP_c_proj.pt")
            .to(torch::kCUDA)
            .to(torch::kHalf);
  }

  void init_intermedia_tensor() {
    // auto attn_fc_output =
    //     torch::ones({batch_size * max_seq_length, d_model}, options_fp16);
    // feed_forward_fc1_weight = torch::ones({d_model, d_intermedia},
    // options_fp16);
    feed_forward_fc1_bias = torch::zeros({d_intermedia}, options_fp16);
    feed_forward_fc1_output =
        torch::zeros({batch_size * max_seq_length, d_intermedia}, options_fp16);
    feed_forward_fc2_bias = torch::zeros({d_model}, options_fp16);
    feed_forward_fc2_output =
        torch::zeros({batch_size * max_seq_length, d_model}, options_fp16);
    feed_forward_fc2_layer_norm_sum =
        torch::zeros({batch_size * max_seq_length,}, options_fp16);
    feed_forward_fc2_layer_norm_sum_x_2 = torch::zeros(
        {batch_size * max_seq_length,}, options_fp16);
    next_attn_layer_norm_output = torch::zeros({batch_size * max_seq_length, d_model}, options_fp16);
  }

  void init_tensor_pointers() {
    // Note, change here
    ptr_residual = input_tensor.data_ptr<at::Half>();
    ptr_input_tensor = input_tensor.data_ptr<at::Half>();
    ptr_feed_forward_fc1_weight = feed_forward_fc1_weight.data_ptr<at::Half>();
    ptr_feed_forward_fc1_bias = feed_forward_fc1_bias.data_ptr<at::Half>();
    ptr_feed_forward_fc1_output = feed_forward_fc1_output.data_ptr<at::Half>();
    ptr_feed_forward_fc2_weight = feed_forward_fc2_weight.data_ptr<at::Half>();
    ptr_feed_forward_fc2_bias = feed_forward_fc2_bias.data_ptr<at::Half>();
    ptr_feed_forward_fc2_output = feed_forward_fc2_output.data_ptr<at::Half>();
    ptr_feed_forward_fc2_layer_norm_sum =  feed_forward_fc2_layer_norm_sum.data_ptr<at::Half>();
    ptr_feed_forward_fc2_layer_norm_sum_x_2 = feed_forward_fc2_layer_norm_sum_x_2.data_ptr<at::Half>();
    ptr_next_attn_layer_norm_output = next_attn_layer_norm_output.data_ptr<at::Half>();
  }

  void torch_forward() {
    // 0. Layer norm
    // t_input_layer_norm = torch::layer_norm(input_tensor, {d_model,});
    // 1. fc1
    t_feed_forward_fc1_output =
        torch::matmul(input_tensor, feed_forward_fc1_weight);
    t_feed_forward_fc1_output += feed_forward_fc1_bias;
    // 2. relu
    t_feed_forward_fc1_output = torch::relu(t_feed_forward_fc1_output);
    // 3. fc2
    t_feed_forward_fc2_output =
        torch::matmul(t_feed_forward_fc1_output, feed_forward_fc2_weight);
    t_feed_forward_fc2_output += feed_forward_fc2_bias;
    // 4. short cut add
    t_feed_forward_fc2_short_cut_output = t_feed_forward_fc2_output + input_tensor;
    // 5. layer norm
    t_feed_forward_fc2_layer_norm = torch::layer_norm(t_feed_forward_fc2_short_cut_output, {d_model,});
    
    t_layer_norm_sum = torch::mean(t_feed_forward_fc2_short_cut_output, {-1, });
  }

  void souffle_forward() {
    fc1_limited_blocks();
    // fc1();
    fc2();
    fused_feed_forward_seq();
    fused_feed_forward_pipelined();
    layer_norm();
  }

  void fc1() {
    // 1. fc1
    void *fused_feed_forward_fc1_kernel_args[] = {
        (void *)&(ptr_feed_forward_fc1_weight), (void *)&(ptr_input_tensor),
        (void *)&(ptr_feed_forward_fc1_output)};
    
    printf("fc1 shared memory %d KB, grid blocks %d\n",
           FeedForwardFC1Params::kSharedMemory / 1024,
           FeedForwardFC1Params::kGridBlocks);
    const void *cuda_kernel_func = (const void *)
        gemm_three_stage<FeedForwardFC1Params::kWarpRowTiles,
                         FeedForwardFC1Params::kWarpColTiles,
                         kHiddenSize * kHiddenDim, kSeqLength, kHiddenDim, 1>;
    checkCuda(
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            cuda_kernel_func),
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize,
            FeedForwardFC1Params::kSharedMemory),
        __LINE__);
    checkCuda(hipLaunchKernel(reinterpret_cast<const void*>(cuda_kernel_func),
                               dim3(FeedForwardFC1Params::kGridBlocks, 1, 1),
                               dim3(FeedForwardFC1Params::kBlockThreads, 1, 1),
                               fused_feed_forward_fc1_kernel_args,
                               FeedForwardFC1Params::kSharedMemory),
              __LINE__);
    hipDeviceSynchronize();
  }

  void fc1_limited_blocks() {
    void *fused_feed_forward_fc1_kernel_args[] = {
        (void *)&(ptr_feed_forward_fc1_weight), (void *)&(ptr_input_tensor),
        (void *)&(ptr_feed_forward_fc1_output)};
    const int feed_forward_fc1_shared_mem = FeedForwardFC1LimitedBlocksParams::kSharedMemory;
    printf("fc1 shared memory %d KB, grid blocks %d\n",
           feed_forward_fc1_shared_mem / 1024,
           FeedForwardFC1LimitedBlocksParams::kGridBlocks);
    const void *cuda_kernel_func =
        (const void *)gemm_three_stage_limited_blocks<
            FeedForwardFC1LimitedBlocksParams::kWarpRowTiles,
            FeedForwardFC1LimitedBlocksParams::kWarpColTiles,
            FeedForwardFC1LimitedBlocksParams::kMTiles /*kMTiles*/,
            FeedForwardFC1LimitedBlocksParams::kNTiles /*kNTiles*/,
            kHiddenSize * kHiddenDim, kSeqLength, kHiddenDim, 1>;
    checkCuda(
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            cuda_kernel_func),
            hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize,
            feed_forward_fc1_shared_mem),
        __LINE__);
    checkCuda(
        hipLaunchKernel(reinterpret_cast<const void*>(
            cuda_kernel_func),
            dim3(FeedForwardFC1LimitedBlocksParams::kGridBlocks, 1, 1),
            dim3(FeedForwardFC1LimitedBlocksParams::kBlockThreads, 1, 1),
            fused_feed_forward_fc1_kernel_args, feed_forward_fc1_shared_mem),
        __LINE__);
    hipDeviceSynchronize();
  }

  void fc2() {
    void *fused_feed_forward_fc2_kernel_args[] = {
        (void *)&(ptr_feed_forward_fc2_weight),
        (void *)&(ptr_feed_forward_fc1_output),
        (void *)&(ptr_feed_forward_fc2_output)};
    
    const int gemm_k6_shared_mem = FeedForwardFC2Params::kSharedMemory;
    const int kGemmK6BlockThreads = 128;
    printf("gemm_k6 shared memory %d KB, grid blocks %d\n",
           gemm_k6_shared_mem / 1024, FeedForwardFC2Params::kGridBlocks);
    checkCuda(hipFuncSetAttribute(
        (const void *)gemm_k6,
        hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize,
        gemm_k6_shared_mem), __LINE__);
    checkCuda(hipLaunchKernel(
        (const void *)gemm_k6, dim3(FeedForwardFC2Params::kGridBlocks, 1, 1),
        dim3(FeedForwardFC2Params::kBlockThreads, 1, 1), fused_feed_forward_fc2_kernel_args,
        gemm_k6_shared_mem), __LINE__);
  }

  void layer_norm() {
    // auto input = torch::ones({batch_size * max_seq_length, d_model},
    //                                       options_fp16);
    // at::Half* ptr_input = input.data_ptr<at::Half>();
    // hipDeviceSynchronize();
    void* kernel_args[] = {
        (void*)&eps, (void*)&gama, (void*)&beta,
        (void*)&(ptr_feed_forward_fc2_output),
        // (void*)&(ptr_input),
        (void*)&(ptr_feed_forward_fc2_layer_norm_sum),
        (void*)&(ptr_feed_forward_fc2_layer_norm_sum_x_2),
        (void*)&(ptr_next_attn_layer_norm_output),
    };
    // checkCuda(hipLaunchKernel(
    //     (const void*)layer_norm_v1<384, 1280>, dim3(kNumberSM, 1, 1),
    //     dim3(kElementwiseBlockThreads, 1, 1), kernel_args));
    // hipDeviceSynchronize();
    // checkCuda(hipLaunchKernel(
    //     (const void*)layer_norm_v2<384, 1280>, dim3(kNumberSM, 1, 1),
    //     dim3(kElementwiseBlockThreads, 1, 1), kernel_args));
    checkCuda(hipFuncSetAttribute(
        (const void*)layer_norm_v3<384, 1280>,
        hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize,
        FeedForwardFC1LimitedBlocksParams::kSharedMemory), __LINE__);
    checkCuda(hipLaunchKernel(
        (const void*)layer_norm_v3<384, 1280>, dim3(kNumberSM, 1, 1),
        dim3(kElementwiseBlockThreads, 1, 1), kernel_args, FeedForwardFC1LimitedBlocksParams::kSharedMemory), __LINE__);
    hipDeviceSynchronize();
  }

  void fused_fc1_fc2_layernorm_relu() {}

  void fused_feed_forward_pipelined() {
    void* fused_feedforward_kernel_args[] = {
        (void *)&(ptr_input_tensor),
        (void *)&(ptr_input_tensor),
        (void *)&(eps), (void *)&(gama), (void *)&(beta),
        (void *)&(ptr_feed_forward_fc1_weight),
        (void *)&(ptr_feed_forward_fc1_output),
        (void *)&(ptr_feed_forward_fc2_weight),
        (void *)&(ptr_feed_forward_fc2_output),
        (void *)&(ptr_feed_forward_fc2_layer_norm_sum),
        (void *)&(ptr_feed_forward_fc2_layer_norm_sum_x_2),
        (void *)&(ptr_next_attn_layer_norm_output)
    };
    const int fused_shared_memory = FeedForwardFC1LimitedBlocksParams::kSharedMemory;
    // std::max(
    //     FeedForwardFC1LimitedBlocksParams::kSharedMemory,
    //     FeedForwardFC2Params::kSharedMemory);
    
    const int fused_grid_blocks = (int)FeedForwardFC1LimitedBlocksParams::kGridBlocks;
    // std::max(
    //     (int)FeedForwardFC1LimitedBlocksParams::kGridBlocks,
    //     (int)FeedForwardFC2Params::kGridBlocks);
    printf("fused_feed_forward shared memory %d KB, grid blocks %d\n",
           fused_shared_memory / 1024, fused_grid_blocks);
    checkCuda(hipFuncSetAttribute(
        (const void *)fused_feed_forwad_pipeline_kernel,
        hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize,
        fused_shared_memory), __LINE__);
    checkCuda(hipLaunchCooperativeKernel((const void *)fused_feed_forwad_pipeline_kernel,
        dim3(fused_grid_blocks, 1, 1),
        dim3(FeedForwardFC1LimitedBlocksParams::kBlockThreads, 1, 1),
        fused_feedforward_kernel_args, fused_shared_memory), __LINE__);
    hipDeviceSynchronize();
  }

  void fused_feed_forward_seq() {
    void* fused_feedforward_kernel_args[] = {
        (void *)&(ptr_input_tensor),
        (void *)&(ptr_input_tensor),
        (void *)&(eps), (void *)&(gama), (void *)&(beta),
        (void *)&(ptr_feed_forward_fc1_weight),
        (void *)&(ptr_feed_forward_fc1_output),
        (void *)&(ptr_feed_forward_fc2_weight),
        (void *)&(ptr_feed_forward_fc2_output),
        (void *)&(ptr_feed_forward_fc2_layer_norm_sum),
        (void *)&(ptr_feed_forward_fc2_layer_norm_sum_x_2),
        (void *)&(ptr_next_attn_layer_norm_output)
    };
    const int fused_shared_memory = FeedForwardFC1LimitedBlocksParams::kSharedMemory;
    // std::max(
    //     FeedForwardFC1LimitedBlocksParams::kSharedMemory,
    //     FeedForwardFC2Params::kSharedMemory);
    
    const int fused_grid_blocks = (int)FeedForwardFC1LimitedBlocksParams::kGridBlocks;
    // std::max(
    //     (int)FeedForwardFC1LimitedBlocksParams::kGridBlocks,
    //     (int)FeedForwardFC2Params::kGridBlocks);
    printf("fused_feed_forward shared memory %d KB, grid blocks %d\n",
           fused_shared_memory / 1024, fused_grid_blocks);
    checkCuda(hipFuncSetAttribute(
        (const void *)fused_feed_forwad_seq_kernel,
        hipFuncAttribute::hipFuncAttributeMaxDynamicSharedMemorySize,
        fused_shared_memory), __LINE__);
    checkCuda(hipLaunchCooperativeKernel((const void *)fused_feed_forwad_seq_kernel,
        dim3(fused_grid_blocks, 1, 1),
        dim3(FeedForwardFC1LimitedBlocksParams::kBlockThreads, 1, 1),
        fused_feedforward_kernel_args, fused_shared_memory), __LINE__);
    hipDeviceSynchronize();
  }

  void print() {
    // printf("feed_forward_fc1_output:");
    // torch::print(this->feed_forward_fc1_output);
    // printf("\nt_feed_forward_fc1_output:");
    // torch::print(this->t_feed_forward_fc1_output);
    // printf("\nfeed_forward_fc2_output:");
    // torch::print(this->feed_forward_fc2_output);
    // printf("\nt_feed_forward_fc2_output:");
    // torch::print(this->t_feed_forward_fc2_output);
    // printf("\nfeed_forward_fc2_short_cut_output:");
    // torch::print(this->t_feed_forward_fc2_short_cut_output);
    printf("\nt_feed_forward_fc2_layer_norm");
    torch::print(t_feed_forward_fc2_layer_norm);
    printf("\nnext_attn_layer_norm_output");
    torch::print(next_attn_layer_norm_output);
    printf("\nlayer_norm_sum:");
    torch::print(t_layer_norm_sum);
    printf("\nour_layer_norm_sum:");
    torch::print(feed_forward_fc2_layer_norm_sum);
    // my_compare(this->feed_forward_fc1_output, this->t_feed_forward_fc1_output, 1.0/16, 1.0/16, kPrintDiff);
    // my_compare(this->feed_forward_fc2_output, this->t_feed_forward_fc2_output, 1.0/16, 1.0/16, kPrintDiff);
    // my_compare(this->feed_forward_fc2_output, this->t_feed_forward_fc2_short_cut_output, 1.0/16, 1.0/16, kPrintDiff);
  }

  std::vector<at::Half *> get_pointers() {
    std::vector<at::Half *> pointers;
    pointers.push_back(ptr_feed_forward_fc1_weight);
    pointers.push_back(ptr_feed_forward_fc1_bias);
    pointers.push_back(ptr_feed_forward_fc1_output);
    pointers.push_back(ptr_feed_forward_fc2_weight);
    pointers.push_back(ptr_feed_forward_fc2_bias);
    pointers.push_back(ptr_feed_forward_fc2_output);

    return pointers;
  }

  const int64_t d_model = num_heads * hidden_size;
  std::string folder_path;
  torch::Tensor input_tensor;
  // Weights
  torch::Tensor input_layer_norm;
  torch::Tensor feed_forward_fc1_weight;
  torch::Tensor feed_forward_fc1_bias;
  torch::Tensor feed_forward_fc1_output;
  torch::Tensor feed_forward_fc2_weight;
  torch::Tensor feed_forward_fc2_bias;
  torch::Tensor feed_forward_fc2_output;
  torch::Tensor feed_forward_fc2_shortcut_output;
  torch::Tensor feed_forward_fc2_layer_norm_sum;
  torch::Tensor feed_forward_fc2_layer_norm_sum_x_2;
  torch::Tensor next_attn_layer_norm_output;
  // Torch output tensors
  torch::Tensor t_input_layer_norm;
  torch::Tensor t_feed_forward_fc1_output;
  torch::Tensor t_feed_forward_fc1_activation_output;
  torch::Tensor t_feed_forward_fc2_output;
  torch::Tensor t_feed_forward_fc2_short_cut_output;
  torch::Tensor t_feed_forward_fc2_layer_norm;
  torch::Tensor t_layer_norm_sum;
  // Pointers
  at::Half *ptr_residual;
  at::Half *ptr_input_tensor;
  at::Half *ptr_feed_forward_fc1_weight;
  at::Half *ptr_feed_forward_fc1_bias;
  at::Half *ptr_feed_forward_fc1_output;
  at::Half *ptr_feed_forward_fc2_weight;
  at::Half *ptr_feed_forward_fc2_bias;
  at::Half *ptr_feed_forward_fc2_output;
  at::Half *ptr_feed_forward_fc2_shortcut_output;
  at::Half *ptr_feed_forward_fc2_layer_norm_sum;
  at::Half *ptr_feed_forward_fc2_layer_norm_sum_x_2;
  at::Half *ptr_next_attn_layer_norm_output;
  half eps = 0.00001, gama = 1, beta = 0;
};

int main(int argc, char *argv[]) {
  std::string folder_path =
      "/home/xiachunwei/Projects/tensor-compiler-gpu/src/operator_fusion/gpt-2/";
//   torch::Tensor feed_forward_input_tensor =
//       torch::ones({384, 20 * 64}, torch::kCUDA).to(torch::kHalf);
  torch::Tensor feed_forward_input_tensor =
      torch_load_tensor(folder_path + "gpt2-torch-data/MLP_input_hidden_states.pt")
          .to(torch::kCUDA)
          .to(torch::kHalf);
  FeedForward<1, 20, 384, 64, 5120> module_feed_forward(
      folder_path, feed_forward_input_tensor);
  module_feed_forward.torch_forward();
  module_feed_forward.souffle_forward();
  module_feed_forward.print();
}
