#include "hip/hip_runtime.h"
// Deal with batch_size < 4
#include "../torch_utils.h"
#include "hip/hip_fp16.h"
#include "torch/all.h"
#include "../../cuda_kernel_utils.h"
#define kBlockSize 256
#define kGridSize 84  // The number of SM on RTX3090 is 84
// #include "kernels/vector_matrix_mul.cu"


__global__ void __launch_bounds__(kBlockSize)
    vector_matrix_mul_kernel(half *__restrict__ input,
                             half *__restrict__ weight,
                             half *__restrict__ output) {
  const int warpIdx = threadIdx.x / 32;
  const int laneIdx = threadIdx.x % 32;
  const int numWarp = kBlockSize / 32;
  const int vectorLength = sizeof(float4) / sizeof(half);
  half8 local_input;
  half8 local_weight;
  const int64_t batch_size = 1;
  const int64_t reduce_dim = 1280;
  const int64_t out_dim = 5120;
  // Iterate over batch_size
  for (int64_t b = 0; b < batch_size; ++b) {
    // Iterate over out_dim
    for (int64_t idx = 0; UPDIV(out_dim, kGridSize * numWarp); ++idx) {
      // Each warp reduce one reduce_dim
      float local_sum = 0;
      const int64_t weight_row_idx =
          (idx * kGridSize * numWarp + blockIdx.x * numWarp + warpIdx);
      // Guard against over indexing
      if (weight_row_idx >= out_dim) break;
#pragma unroll
      for (int64_t k = 0; k < reduce_dim; k += (warpSize * vectorLength)) {
        const int64_t col_idx = k + laneIdx * vectorLength;
        // Guard against over indexing
        if (col_idx >= reduce_dim) break;
        *((float4 *)&local_input) =
            *((float4 *)&(input[(b * reduce_dim + col_idx)]));
        *((float4 *)&local_weight) =
            *((float4 *)&(weight[(weight_row_idx * reduce_dim + col_idx)]));
        float2 tmp;
        tmp = __half22float2(__hmul2(half2(local_input.data[0], local_input.data[1]), half2(local_weight.data[0], local_weight.data[1])));
        local_sum += (tmp.x + tmp.y);
        tmp = __half22float2(__hmul2(half2(local_input.data[2], local_input.data[3]), half2(local_weight.data[2], local_weight.data[3])));
        local_sum += (tmp.x + tmp.y);
        tmp = __half22float2(__hmul2(half2(local_input.data[4], local_input.data[5]), half2(local_weight.data[4], local_weight.data[5])));
        local_sum += (tmp.x + tmp.y);
        tmp = __half22float2(__hmul2(half2(local_input.data[6], local_input.data[7]), half2(local_weight.data[6], local_weight.data[7])));
        local_sum += (tmp.x + tmp.y);
      }
      // Reduce within warp
      local_sum = warpReduceSum(local_sum);
      // Write to output
      if (laneIdx == 0) {
        output[b * out_dim + weight_row_idx] = __float2half(local_sum);
      }
    }
  }
}


int main(int argc, char* argv[]) {
  // Load weight
  std::string folder_path =
      "/home/xiachunwei/Projects/tensor-compiler-gpu/src/operator_fusion/gpt-2/";
  // Shape (1280, 5120)
  torch::Tensor attn_fc_weight = torch_load_tensor(folder_path + "gpt2-torch-data/MLP_c_fc.pt")
            .to(torch::kCUDA)
            .to(torch::kHalf);
  const int reduce_dim = attn_fc_weight.sizes()[0];
  const int out_dim = attn_fc_weight.sizes()[1];
  const int batch_size = 1;

  auto src =
      torch::ones({batch_size, reduce_dim}, torch::kHalf).to(torch::kCUDA);
  auto output =
      torch::empty({batch_size, out_dim}, options_fp16).to(torch::kCUDA);
  auto permuted_attn_fc_weight = torch::permute(attn_fc_weight, {1, 0}).contiguous();

  // Declare pointers
  auto d_ptr_input = src.data_ptr<at::Half>();
  // Note, need to permute to make the reduction dimension contiguous
  auto d_ptr_weight = permuted_attn_fc_weight.data_ptr<at::Half>();
  auto cpu_permuted_attn_fc_weight = permuted_attn_fc_weight.to(torch::kCPU);
  auto d_ptr_output = output.data_ptr<at::Half>();
  // Check correctness
  auto torch_output = torch::mm(
      src, attn_fc_weight);  // (m, k) * (k, n) = (m, n)
  hipDeviceSynchronize();

  void* args[] = {
    (void**)&d_ptr_input, (void**)&d_ptr_weight, (void**)&d_ptr_output
  };
    // Launch kernel
  hipLaunchKernel((void*)vector_matrix_mul_kernel, dim3(kGridSize, 1, 1), 
    dim3(kBlockSize, 1, 1), args);

  hipDeviceSynchronize();
  
  printf("src\n");
  torch::print(src);
  printf("permuted_attn_fc_weight\n");
  torch::print(permuted_attn_fc_weight);
  printf("torch_output\n");
  torch::print(torch_output);
  printf("output\n");
  torch::print(output);
  printf("%d\n", torch::allclose(output, torch_output, 1e-2, 1e-3));
  return 0;
}
