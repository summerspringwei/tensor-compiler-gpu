#include "hip/hip_runtime.h"

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <cstdlib>
#include <time.h> 

#include "../../utils.h"
#include "../../cuda_utils.h"

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)


void init_inputs_and_weights(float* input, float* weight1, float* bias1, float* weight2, float* bias2,
  int in_channels, int height, int width, int out_channels_1, int out_channels_2){
  init_values(input, {height, width, in_channels}, 1);
  init_values(weight1, {out_channels_1, in_channels}, 0.01);
  init_values(bias1, {out_channels_1}, 1);
  init_values(weight2, {out_channels_1, out_channels_2}, 0.01);
  init_values(bias2, {out_channels_2}, 1);
}

extern "C" __global__ void __launch_bounds__(256) fused_micro_operators(
  float* input, float* weight1, float* bias1, float* weight2, float* bias2, float* output);

int main() {
  // Declare size
  const int batch = 1;
  const int in_channels = 480, height = 14, width=14;
  const int out_channels_1 = 20, out_channels_2 = 480;
  const int input_size = batch*height*width*in_channels;
  const int weight1_size = in_channels * out_channels_1;
  const int weight2_size = out_channels_1 * out_channels_2;
  const int output_size = out_channels_2;


  // Declare arrays
  float *input = new float[input_size];
  float *intermedia_reduce_mean = new float[in_channels];
  float *weight1 = new float[weight1_size];
  float *bias1 = new float[out_channels_1];
  float *intermedia_output1 = new float[out_channels_1];
  float *weight2 = new float[weight2_size];
  float *bias2 = new float[out_channels_2];
  float *output = new float[output_size];
  float *ori_output = new float[output_size];

  float* d_input = NULL, *d_intermedia_reduce_mean=NULL,
    *d_weight1 = NULL, *d_bias1=NULL, *d_intermedia_output1=NULL,
    *d_weight2=NULL, *d_bias2 = NULL, *d_output = NULL, *d_ori_output = NULL;
  
  // Allocate space on device
  hipError_t err = hipSuccess;
  err = hipMalloc((void **)&d_input, sizeof(float)*input_size);
  err = hipMalloc((void **)&d_intermedia_reduce_mean, sizeof(float)*in_channels);
  err = hipMalloc((void **)&d_weight1, sizeof(float)*weight1_size);
  err = hipMalloc((void **)&d_bias1, sizeof(float)*out_channels_1);
  err = hipMalloc((void **)&d_intermedia_output1, sizeof(float)*out_channels_1);
  err = hipMalloc((void **)&d_weight2, sizeof(float)*weight2_size);
  err = hipMalloc((void **)&d_bias2, sizeof(float)*out_channels_2);
  err = hipMalloc((void **)&d_output, sizeof(float)*output_size);
  err = hipMalloc((void **)&d_ori_output, sizeof(float)*output_size);

  // Check error
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Copy data and convert data type
  init_inputs_and_weights(input, weight1, bias1, weight2, bias2,
    in_channels, height, width, out_channels_1, out_channels_2);
  hipMemcpy(d_input, input, sizeof(float)*input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_weight1, weight1, sizeof(float)*weight1_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias1, bias1, sizeof(float)*out_channels_1, hipMemcpyHostToDevice);
  hipMemcpy(d_weight2, weight2, sizeof(float)*weight2_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias2, bias2, sizeof(float)*out_channels_2, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // Warm up
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  fused_micro_operators<<<dim3(1, 1, 1), dim3(256, 1, 1)>>>(d_input, d_weight1, d_bias1, d_weight2, d_bias2, d_output);
  hipDeviceSynchronize();
  // err = hipMemcpy(ori_output, d_ori_output, sizeof(float)*output_size, hipMemcpyDeviceToHost);
  err = hipMemcpy(output, d_output, sizeof(float)*output_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess){
    fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Benchmark
  const int loop = 100;
  float ms = 0, sum = 0, min = 10000, max=0;
  // 1. For original pointwise conv
  for(int i=0; i<loop; ++i){
    checkCuda( hipEventRecord(startEvent,0) );
    
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    sum += ms;
    ms > max? max=ms: 0;
    ms < min? min=ms: 0;
  }printf("Before fuse avg time %f, min %f, max %f\n", sum / loop, min, max);
  sum = 0, min = 10000, max=0;;
  for(int i=0; i<loop; ++i){
    checkCuda( hipEventRecord(startEvent,0) );
    
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    sum += ms;
    ms > max? max=ms: 0;
    ms < min? min=ms: 0;
  }printf("Before fuse avg time %f, min %f, max %f\n", sum / loop, min, max);
  
  ms = 0, sum = 0, min = 10000, max=0;
  for(int i=0; i<loop; ++i){
    checkCuda( hipEventRecord(startEvent,0) );
    fused_micro_operators<<<dim3(1, 1, 1), dim3(256, 1, 1)>>>(d_input, d_weight1, d_bias1, d_weight2, d_bias2, d_output);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    sum += ms;
    ms > max? max=ms: 0;
    ms < min? min=ms: 0;
  }printf("After fuse avg time %f, min %f, max %f\n", sum / loop, min, max);
  sum = 0;
  for(int i=0; i<loop; ++i){
    checkCuda( hipEventRecord(startEvent,0) );
    fused_micro_operators<<<dim3(1, 1, 1), dim3(256, 1, 1)>>>(d_input, d_weight1, d_bias1, d_weight2, d_bias2, d_output);
    checkCuda( hipEventRecord(stopEvent,0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    sum += ms;  
    ms > max? max=ms: 0;
    ms < min? min=ms: 0;
  }printf("After fuse avg time %f, min %f, max %f\n", sum / loop, min, max);
  
  
  // Print result
  printf("outputs:->\n");
  bool equal = true;
  for (int oc = 0; oc < out_channels_2; ++oc) {
    printf("%.6f ", output[oc]);
  }printf("\n");
  if(equal){
    printf("Check passed\n");
  }else{
    printf("Check failed\n");
  }
  
  // Free
  hipFree(d_input);
  hipFree(d_intermedia_reduce_mean);
  hipFree(d_weight1);
  hipFree(d_bias1);
  hipFree(d_intermedia_output1);
  hipFree(d_weight2);
  hipFree(d_bias2);
  hipFree(d_output);
  hipFree(d_ori_output);
  delete []input;
  delete []intermedia_reduce_mean;
  delete []weight1;
  delete []bias1;
  delete []intermedia_output1;
  delete []weight2;
  delete []bias2;
  delete []output;
  delete []ori_output;
  return 0;
}
