#include "hip/hip_runtime.h"


#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>

#include "fused_pointwise_pointwise.h"
#include "auto_scheduler_codegen/pointwise_112_112_16_32.h"
#include "auto_scheduler_codegen/pointwise_112_112_32_96.h"

#include "../../utils.h"
#include "../../cuda_utils.h"

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

void init_inputs_and_weights(float* input, float* weight1, float* bias1, float* weight2, float* bias2,
  int in_channels, int height, int width, int out_channels_1, int out_channels_2){
  init_values(input, {height, width, in_channels}, 1, 1);
  init_values(weight1, {in_channels, out_channels_1}, 1, 1);
  init_values(bias1, {out_channels_1}, 1);
  init_values(weight2, {out_channels_1, out_channels_2}, 1, 1);
  init_values(bias2, {out_channels_2}, 1);
}


int main() {
  // Declare size
  const int height = 112, width = 112, in_channels=32, out_channels_1=16, out_channels_2 = 96, block_size = 256, num_blocks = height * width / 32;
  // const int kernel_height = 1, kernel_width = 1;
  const int input_size = in_channels * height * width;
  const int weight1_size = in_channels * out_channels_1;
  const int weight2_size = out_channels_1 * out_channels_2;
  const int tmp_output_size = height * width * out_channels_1;
  const int output_size = height * width * out_channels_2;

  // Declare arrays
  float *input = new float[input_size];
  float *pw_weight1 = new float[weight1_size];
  float *bias1 = new float[out_channels_1];
  float *pw_weight2 = new float[weight2_size];
  float* bias2 = new float[out_channels_2];
  float *tmp_output = new float[tmp_output_size];
  float *output = new float[output_size];
  float *cpu_output = new float[output_size];
  float* d_input = NULL, *d_pw_weight1 = NULL, *d_pw_weight2=NULL, *d_tmp_output=NULL, *d_output = NULL, *d_ori_output=NULL;
  

  // Allocate space on device
  hipError_t err = hipSuccess;
  err = hipMalloc((void **)&d_input, sizeof(float)*input_size);
  err = hipMalloc((void **)&d_pw_weight1, sizeof(float)*weight1_size);
  err = hipMalloc((void **)&d_pw_weight2, sizeof(float)*weight2_size);
  err = hipMalloc((void **)&d_tmp_output, sizeof(float)*tmp_output_size);
  err = hipMalloc((void **)&d_output, sizeof(float)*output_size);
  err = hipMalloc((void **)&d_ori_output, sizeof(float)*output_size);

  // Check error
  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  
  // Copy data and convert data type
  init_inputs_and_weights(input, pw_weight1, bias1, pw_weight2, bias2, in_channels, height, width, out_channels_1, out_channels_2);

  // CPU implementation to check result
  pointwise_conv(input, pw_weight1, tmp_output, height, width, in_channels, out_channels_1);
  pointwise_conv(tmp_output, pw_weight2, cpu_output, height, width, out_channels_1, out_channels_2);

  hipMemcpy(d_input, input, sizeof(float)*input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_pw_weight1, pw_weight1, sizeof(float)*weight1_size, hipMemcpyHostToDevice);
  hipMemcpy(d_pw_weight2, pw_weight2, sizeof(float)*weight2_size, hipMemcpyHostToDevice);
  hipDeviceSynchronize();

  // Warm up
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  fused_pointwise_pointwise<num_blocks, block_size, height, width, in_channels, out_channels_1, out_channels_2>\
    <<<dim3(num_blocks,1,1), dim3(block_size,1,1)>>>(d_input, d_pw_weight1, d_pw_weight2, d_output);
  pointwise_112_112_16_32<<<dim3(784,1,1),  dim3(32,1,1)>>>(d_input, d_pw_weight1, d_tmp_output);
  pointwise_112_112_32_96<<<dim3(784,1,1),  dim3(32,1,1)>>>(d_tmp_output, d_pw_weight2, d_ori_output);
  err = hipMemcpy(output, d_output, sizeof(float)*output_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess){
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Benchmark
  const int round_cout = 2, loop = 10000;
  float ms = 0, sum = 0;
  // 1. For original pointwise conv
  for(int round =0; round<round_cout; ++round){
    ms = 0, sum = 0;
    for(int i=0; i<loop; ++i){
      checkCuda( hipEventRecord(startEvent,0) );
      pointwise_112_112_16_32<<<dim3(784,1,1),  dim3(32,1,1)>>>(d_input, d_pw_weight1, d_tmp_output);
      pointwise_112_112_32_96<<<dim3(784,1,1),  dim3(32,1,1)>>>(d_tmp_output, d_pw_weight2, d_ori_output);
      checkCuda( hipEventRecord(stopEvent,0) );
      checkCuda( hipEventSynchronize(stopEvent) );
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      sum += ms;
    }printf("Before fuse avg time %f\n", sum / loop);
  }
  
  for(int round =0; round<round_cout; ++round){
    ms = 0, sum = 0;
    for(int i=0; i<loop; ++i){
      checkCuda( hipEventRecord(startEvent,0) );
      fused_pointwise_pointwise<num_blocks, block_size, height, width, in_channels, out_channels_1, out_channels_2>\
      <<<dim3(num_blocks,1,1), dim3(block_size,1,1)>>>(d_input, d_pw_weight1, d_pw_weight2, d_output);
      checkCuda( hipEventRecord(stopEvent,0) );
      checkCuda( hipEventSynchronize(stopEvent) );
      checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
      sum += ms;
    }printf("After fuse avg time %f\n", sum / loop);
  }
  
  check_equal(output, cpu_output, height, width, out_channels_2);


  // Free
  hipFree(d_input);
  hipFree(d_pw_weight1);
  hipFree(d_pw_weight2);
  hipFree(d_output);
  hipFree(d_ori_output);
  hipFree(d_tmp_output);
  delete[] input;
  delete[] pw_weight1;
  delete[] pw_weight2;
  delete[] output;
  delete[] cpu_output;
  return 0;
}
