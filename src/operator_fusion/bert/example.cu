#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cuda/pipeline>
 
template <typename T>
__global__ void example_kernel(T * global1, T * global2, size_t subset_count)
{
    extern __shared__ T s[];
    constexpr unsigned stages_count = 2;
    auto group = cooperative_groups::this_thread_block();
    T * shared[stages_count] = { s, s + 2 * group.size() };
 
    // Create a synchronization object (cuda::pipeline)
    __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block, stages_count> shared_state;
    auto pipeline = cuda::make_pipeline(group, &shared_state);
 
    size_t fetch;
    size_t subset;
    for (subset = fetch = 0; subset < subset_count; ++subset) {
        // Fetch ahead up to stages_count subsets
        for (; fetch < subset_count && fetch < (subset + stages_count); ++fetch ) {
            pipeline.producer_acquire();
            cuda::memcpy_async(group, shared[fetch % 2],
                               &global1[fetch * group.size()], sizeof(T) * group.size(), pipeline);
            cuda::memcpy_async(group, shared[fetch % 2] + group.size(),
                               &global2[fetch * group.size()], sizeof(T) * group.size(), pipeline);
            pipeline.producer_commit(); // Commit the fetch-ahead stage
        }
        pipeline.consumer_wait(); // Wait for ‘subset’ stage to be available
 
        compute(shared[subset % 2]);
 
        pipeline.consumer_release();
    }
}


template <size_t stages_count = 2 /* Pipeline with stages_count stages */>
__global__ void with_staging_unified(int* global_out, int const* global_in, size_t size, size_t batch_sz) {
    auto grid = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();
    assert(size == batch_sz * grid.size()); // Assume input size fits batch_sz * grid_size

    extern __shared__ int shared[]; // stages_count * block.size() * sizeof(int) bytes
    size_t shared_offset[stages_count];
    for (int s = 0; s < stages_count; ++s) shared_offset[s] = s * block.size();

    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;
    auto pipeline = cuda::make_pipeline(block, &shared_state);

    auto block_batch = [&](size_t batch) -> int {
        return block.group_index().x * block.size() + grid.size() * batch;
    };

    // compute_batch: next batch to process
    // fetch_batch:  next batch to fetch from global memory
    for (size_t compute_batch = 0, fetch_batch = 0; compute_batch < batch_sz; ++compute_batch) {
        // The outer loop iterates over the computation of the batches
        for (; fetch_batch < batch_sz && fetch_batch < (compute_batch + stages_count); ++fetch_batch) {
            // This inner loop iterates over the memory transfers, making sure that the pipeline is always full
            pipeline.producer_acquire();
            size_t shared_idx = fetch_batch % stages_count;
            size_t batch_idx = fetch_batch;
            size_t block_batch_idx = block_batch(batch_idx);
            cuda::memcpy_async(block, shared + shared_offset[shared_idx], global_in + block_batch_idx, sizeof(int) * block.size(), pipeline);
            pipeline.producer_commit();
        }
        pipeline.consumer_wait();
        int shared_idx = compute_batch % stages_count;
        int batch_idx = compute_batch;
        compute(global_out + block_batch(batch_idx), shared + shared_offset[shared_idx]);
        pipeline.consumer_release();
    }
}


#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

__device__ void compute(int* global_out, int const* shared_in);
__global__ void with_staging(int* global_out, int const* global_in, size_t size, size_t batch_sz) {
    auto grid = cooperative_groups::this_grid();
    auto block = cooperative_groups::this_thread_block();
    assert(size == batch_sz * grid.size()); // Assume input size fits batch_sz * grid_size

    constexpr size_t stages_count = 2; // Pipeline with two stages
    // Two batches must fit in shared memory:
    extern __shared__ int shared[];  // stages_count * block.size() * sizeof(int) bytes
    size_t shared_offset[stages_count] = { 0, block.size() }; // Offsets to each batch

    // Allocate shared storage for a two-stage cuda::pipeline:
    __shared__ cuda::pipeline_shared_state<
        cuda::thread_scope::thread_scope_block,
        stages_count
    > shared_state;
    auto pipeline = cuda::make_pipeline(block, &shared_state);

    // Each thread processes `batch_sz` elements.
    // Compute offset of the batch `batch` of this thread block in global memory:
    auto block_batch = [&](size_t batch) -> int {
      return block.group_index().x * block.size() + grid.size() * batch;
    };

    // Initialize first pipeline stage by submitting a `memcpy_async` to fetch a whole batch for the block:
    if (batch_sz == 0) return;
    pipeline.producer_acquire();
    cuda::memcpy_async(block, shared + shared_offset[0], global_in + block_batch(0), sizeof(int) * block.size(), pipeline);
    pipeline.producer_commit();

    // Pipelined copy/compute:
    for (size_t batch = 1; batch < batch_sz; ++batch) {
        // Stage indices for the compute and copy stages:
        size_t compute_stage_idx = (batch - 1) % 2;
        size_t copy_stage_idx = batch % 2;

        size_t global_idx = block_batch(batch);

        // Collectively acquire the pipeline head stage from all producer threads:
        pipeline.producer_acquire();

        // Submit async copies to the pipeline's head stage to be
        // computed in the next loop iteration
        cuda::memcpy_async(block, shared + shared_offset[copy_stage_idx], global_in + global_idx, sizeof(int) * block.size(), pipeline);
        // Collectively commit (advance) the pipeline's head stage
        pipeline.producer_commit();

        // Collectively wait for the operations commited to the
        // previous `compute` stage to complete:
        pipeline.consumer_wait();

        // Computation overlapped with the memcpy_async of the "copy" stage:
        compute(global_out + global_idx, shared + shared_offset[compute_stage_idx]);

        // Collectively release the stage resources
        pipeline.consumer_release();
    }

    // Compute the data fetch by the last iteration
    pipeline.consumer_wait();
    compute(global_out + block_batch(batch_sz-1), shared + shared_offset[(batch_sz - 1) % 2]);
    pipeline.consumer_release();
}