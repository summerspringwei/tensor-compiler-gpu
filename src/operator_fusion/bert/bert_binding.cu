// #include  <pybind11/pybind11.h>

// int add(int i, int j) {
// return i + j;
// }
// PYBIND11_MODULE(bert, m) {
// m.doc() = "pybind11 example plugin"; // optional module docstring
// m.def("add", &add, "A function that adds two numbers");
// }

#include <iostream>

#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>

#include <torch/extension.h>

#include "bert_fused_fc_fc.h"
#include "bert_query_key_matmul_softmax.h"

torch::Tensor d_sigmoid(torch::Tensor z) {
  auto s = torch::sigmoid(z);
  return (1 - s) * s;
}

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")


void check_compatability(int numThreads, void* cuda_kernel){
  int dev = 0;
  int supportsCoopLaunch = 0;
  hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
  if(supportsCoopLaunch){
    printf("Device support CoopLaunch\n");
  }
  hipDeviceProp_t deviceProp; \
  hipGetDeviceProperties(&deviceProp, dev); \
  int numBlocksPerSm;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, cuda_kernel, numThreads, 0); 
  printf("fused_fc_fc: OccupancyMaxActiveBlocksPerMultiprocessor: %d, multiProcessorCount: %d\n", numBlocksPerSm, deviceProp.multiProcessorCount);
}


template<int64_t batch_size, int64_t num_heads, int64_t max_seq_length, int64_t hidden_size>
torch::Tensor fused_query_key_matmul_softmax(torch::Tensor query, torch::Tensor key) {
  // Check input
  CHECK_CUDA(query);
  CHECK_CUDA(key);
  // assert(query.dense_dim()==3 && key.dense_dim()==3);
  assert(query.size(0)==batch_size*num_heads && query.size(1)==max_seq_length && query.size(2)==hidden_size);
  assert(key.size(0)==batch_size*num_heads && key.size(1)==max_seq_length && key.size(2)==hidden_size);
  
  // Check compatability
  check_compatability(32, (void*)fused_query_key_matmul_softmax_v3);

  auto options = torch::TensorOptions()
    .dtype(torch::kFloat16)
    .layout(torch::kStrided)
    .device(torch::kCUDA, 0)
    .requires_grad(false);
  auto sum = torch::zeros({batch_size*num_heads, max_seq_length}, options);
  auto output = torch::zeros({batch_size*num_heads, max_seq_length, max_seq_length}, options);
  // void *kernel_args[] = { (void *)(query.data_ptr()), (void *)(key.data_ptr()), (void *)(output.data_ptr()), (void *)(sum.data_ptr()) };
  at::Half* ptr_query = query.data<at::Half>();
  at::Half* ptr_key = key.data<at::Half>();
  at::Half* ptr_output = output.data<at::Half>();
  at::Half* ptr_sum = sum.data<at::Half>();
  void *kernel_args[] = { (void *)(&ptr_query), (void *)(&ptr_key), (void *)(&ptr_output), (void *)(&ptr_sum) };
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(output.type(), "fused_query_key_matmul_softmax", [&]{
    checkCuda(hipLaunchCooperativeKernel((void*)fused_query_key_matmul_softmax_v3, dim3(4, 4,12), dim3(32,1,1), kernel_args, 8704*sizeof(half)));
  });
  hipDeviceSynchronize();
  return output;
}

template<int64_t batch_size, int64_t num_heads, int64_t max_seq_length, int64_t hidden_size, int64_t dim_feedforward>
std::vector<torch::Tensor> fused_feed_forward(torch::Tensor src, torch::Tensor weight1, torch::Tensor weight2){
  // Check input
  CHECK_CUDA(src);
  CHECK_CUDA(weight1);
  CHECK_CUDA(weight2);
  assert(src.size(0)==batch_size*max_seq_length && src.size(1)==num_heads*hidden_size);
  assert(weight1.size(0)==dim_feedforward && weight1.size(1)==num_heads*hidden_size);
  assert(weight2.size(0)==num_heads*hidden_size && weight2.size(1)==dim_feedforward);

  check_compatability(128, (void*)fused_fc_fc_v2);
  // Create outputs
  auto options_fp16 = torch::TensorOptions()
    .dtype(torch::kFloat16)
    .layout(torch::kStrided)
    .device(torch::kCUDA, 0)
    .requires_grad(false);
  auto output1 = torch::zeros({batch_size*max_seq_length, dim_feedforward}, options_fp16);
  auto output2 = torch::zeros({batch_size*max_seq_length, num_heads*hidden_size}, options_fp16);
  auto options_fp32 = torch::TensorOptions()
    .dtype(torch::kFloat32)
    .layout(torch::kStrided)
    .device(torch::kCUDA, 0)
    .requires_grad(false);
  auto sum = torch::zeros({batch_size*max_seq_length,}, options_fp32);
  auto variance = torch::zeros({batch_size*max_seq_length,}, options_fp32);

  at::Half* ptr_src = src.data<at::Half>();
  at::Half* ptr_weight1 = weight1.data<at::Half>();
  at::Half* ptr_output1 = output1.data<at::Half>();
  at::Half* ptr_weight2 = weight2.data<at::Half>();
  at::Half* ptr_output2 = output2.data<at::Half>();
  float* ptr_sum = sum.data<float>();
  float* ptr_variance = variance.data<float>();
  half eps = 0.00001, gama=1, beta=0;
  // fused_fc_fc_v2(half *__restrict__ x, half *__restrict__ placeholder,
  //               half *__restrict__ T_dense, half *__restrict__ placeholder2,
  //               half *__restrict__ T_dense2, half* sum, half* variance, half eps, half gama, half beta) 
  

  void *fused_kernel_args[] = { (void *)&(ptr_src), (void *)&(ptr_weight1), 
    (void *)&(ptr_output1), (void *)&(ptr_weight2), (void *)&(ptr_output2), 
    (void *)&(ptr_sum), (void *)&(ptr_variance), (void*)&eps, (void*)&gama, (void*)&beta};
  
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(output2.type(), "fused_feed_forward", [&]{
    checkCuda(hipLaunchCooperativeKernel((void*)fused_fc_fc_v2, dim3(192, 1, 1), dim3(128, 1, 1), fused_kernel_args, 13056 * sizeof(half)));
  });
  
  return {output2, sum, variance};
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("d_sigmoid", &d_sigmoid, "d_sigmoid function");
  m.def("fused_query_key_matmul_softmax", &fused_query_key_matmul_softmax<1, 12, 128, 64>, 
    "bert fused_query_key_matmul_softmax with num_heads=12, max_seq_length=128, hidden_size=64");
  m.def("fused_feed_forward", &fused_feed_forward<1, 12, 128, 64, 3072>, 
    "bert fused_feed_forward with num_heads=12, max_seq_length=128, hidden_size=64, dim_feedforward=3072");
}
