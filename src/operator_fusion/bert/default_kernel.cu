#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>

#include "../../utils.h"
#include "../../cuda_utils.h"


int main(){
	const int qeury_size=98304;	half *query = new half[qeury_size];
	const int key_size=98304;	half *key = new half[key_size];
	const int output_size=196608;	half *output = new half[output_size];
	const int sum_size=1536;	half *sum = new half[sum_size];

	hipError_t err = hipSuccess;
	half *d_qeury=NULL;
	half *d_key=NULL;
	half *d_output=NULL;
	half *d_sum=NULL;
	err=hipMalloc((void **)&d_qeury, sizeof(half)*qeury_size);
	err=hipMalloc((void **)&d_key, sizeof(half)*key_size);
	err=hipMalloc((void **)&d_output, sizeof(half)*output_size);
	err=hipMalloc((void **)&d_sum, sizeof(half)*sum_size);

	hipMemcpy(d_qeury, query, sizeof(half)*qeury_size, hipMemcpyHostToDevice);
	hipMemcpy(d_key, key, sizeof(half)*key_size, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	hipMemcpy(output, d_output, sizeof(half)*output_size, hipMemcpyDeviceToHost);
	hipMemcpy(sum, d_sum, sizeof(half)*sum_size, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

  if (err != hipSuccess){
    fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  	delete[] query;
	delete[] key;
	delete[] output;
	delete[] sum;
	hipFree(d_qeury);
	hipFree(d_key);
	hipFree(d_output);
	hipFree(d_sum);
	return 0;
}