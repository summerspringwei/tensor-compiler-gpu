#include "hip/hip_runtime.h"

#include <cstdlib>
#include <iostream>
#include <vector>
#include <chrono>
#include <sstream>

#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../../cuda_utils.h"
#include "lstm_reuse_shared_memory.h"
#include "lstm_utils.h"


// kNumGatePart=4
// #define LSTM_DEV_FUNC  { \
//     hipLaunchCooperativeKernel((void*)lstm_reuse_shared_memory_v6<1, 10, 256, 100>, dim3(320, 1, 1), dim3(256, 1, 1), encoder_kernelArgs, 48*1024);};

// #define LSTM_DEV_FUNC  { \
//     hipLaunchCooperativeKernel((void*)lstm_reuse_shared_memory_v8<1, 10, 256, 100>, dim3(320, 1, 1), dim3(32, 8, 1), encoder_kernelArgs, 32*1024);};

// #define LSTM_DEV_FUNC  { \
//     hipLaunchCooperativeKernel((void*)lstm_reuse_shared_memory_v9<1, 10, 256, 100>, dim3(320, 1, 1), dim3(32, 8, 1), encoder_kernelArgs, 48*1024);};

#define LSTM_DEV_FUNC  { \
lstm_reuse_shared_memory_v9_block_sync<1, 10, 256, 100><<<dim3(320, 1, 1), dim3(32, 8, 1), 48*1024>>>( \
    (lstm_data.d_inputs_timestep), (lstm_data.d_outputs_timestep), \
    (lstm_data.d_c_wavefront), (lstm_data.d_h_wavefront), (lstm_data.d_input_wavefront), \
    (lstm_data.d_weight_input_wavefront), (lstm_data.d_weight_state_wavefront), (lstm_data.d_bias), \
    (lstm_data.d_output_buffer) ); \
    };

#define CUDA_CHECK_RESULT if (result != hipSuccess) \
    { \
        const char* msg = hipGetErrorString(result); \
        std::stringstream safe_call_ss; \
        safe_call_ss << "\nerror: " << " failed with error" \
                    << "\nfile: " << __FILE__ << "\nline: " << __LINE__ << "\nmsg: " << msg; \
        throw std::runtime_error(safe_call_ss.str()); \
    };


void benchmark_lstm(int argc, char** argv){
    const int batch = 1;
    const int num_layer = 10, num_timestep = 100, num_hidden = 256;
    
    int steps = 100;
    if(argc > 1){
        steps = atoi(argv[1]);
    }
    
    int dev = 0;
    int supportsCoopLaunch = 0;
    hipDeviceGetAttribute(&supportsCoopLaunch, hipDeviceAttributeCooperativeLaunch, dev);
    if(supportsCoopLaunch){
        printf("Device support CoopLaunch\n");
    }
    auto lstm_data = create_lstm_data(batch, num_layer, num_hidden, num_timestep);

    // int* d_arr_sync=nullptr;
    // hipMalloc((void**)&d_arr_sync, 8*num_layer*num_layer*sizeof(int));
    // hipMemset(d_arr_sync, 0, 8*num_layer*num_layer*sizeof(int));
    // Set shared memory for SM
    // int maxbytes = 1024*64;
    // hipFuncSetAttribute((void*)lstm_reuse_shared_memory_v9<1, 10, 256, 100>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    // int carveout = 50; // prefer shared memory capacity 50% of maximum
    // Named Carveout Values:
    // carveout = cudaSharedmemCarveoutDefault;   //  (-1)
    // carveout = cudaSharedmemCarveoutMaxL1;     //   (0)
    // auto carveout = cudaSharedmemCarveoutMaxShared; // (100)
    // hipFuncSetAttribute(reinterpret_cast<const void*>(lstm_wavefront_magic), hipFuncAttributePreferredSharedMemoryCarveout, carveout);
    int numThreads = 64*4, numBlocksPerSm=0; \
    hipDeviceProp_t deviceProp; \
    hipGetDeviceProperties(&deviceProp, dev); \
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, (void*)lstm_reuse_shared_memory_v9_block_sync<1, 10, 256, 100>, numThreads, 0); \
    printf("OccupancyMaxActiveBlocksPerMultiprocessor: %d, multiProcessorCount: %d\n", numBlocksPerSm, deviceProp.multiProcessorCount);\
    void *encoder_kernelArgs[] = { (void *)&(lstm_data.d_inputs_timestep), (void *)&(lstm_data.d_outputs_timestep), \
        (void *)&(lstm_data.d_c_wavefront), (void *)&(lstm_data.d_h_wavefront), (void *)&(lstm_data.d_input_wavefront), \
        (void *)&(lstm_data.d_weight_input_wavefront), (void *)&(lstm_data.d_weight_state_wavefront), (void *)&(lstm_data.d_bias), \
        (void *)&(lstm_data.d_output_buffer)
        };
    
    LSTM_DEV_FUNC
    hipDeviceSynchronize();
    
    std::vector<float> lstm_output_timestep(batch * num_hidden * num_timestep);
    checkCuda(hipMemcpy(lstm_output_timestep.data(), lstm_data.d_outputs_timestep, sizeof(float) * lstm_output_timestep.size() , hipMemcpyDeviceToHost));
    std::vector<float> c_state_timestep(batch * num_layer * num_hidden);
    std::vector<float> h_state_timestep(batch * num_layer * num_hidden);
    checkCuda(hipMemcpy(c_state_timestep.data(), lstm_data.d_c_wavefront, sizeof(float) * c_state_timestep.size() , hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(h_state_timestep.data(), lstm_data.d_h_wavefront, sizeof(float) * h_state_timestep.size() , hipMemcpyDeviceToHost));
    
    // printf("c_state\n");
    // for(int i=0;i<num_layer; ++i){
    //     for(int j=0;j<num_hidden;++j){
    //         printf("%.3f ", c_state_timestep[i*num_hidden + j]);
    //     }printf("\n");
    // }
    // printf("h_state\n");
    // for(int i=0;i<num_layer; ++i){
    //     for(int j=0;j<num_hidden;++j){
    //         printf("%.3f ", h_state_timestep[i*num_hidden + j]);
    //     }printf("\n");
    // }
    // printf("lstm_output_timestep\n");
    // for(int i=0;i<num_timestep; ++i){
    //     printf("step: %d\n", i);
    //     for(int j=0;j<num_hidden;++j){
    //         printf("%.3f ", lstm_output_timestep[i*num_hidden + j]);
    //     }printf("\n");
    // }
    auto result = hipGetLastError();
    CUDA_CHECK_RESULT
    return;
    // Warm up
    for (int i=0; i<steps; i++) {
        LSTM_DEV_FUNC
        hipDeviceSynchronize();
    }
    result = hipGetLastError();                                                   
    CUDA_CHECK_RESULT
     
    // GPU time measurement
    float ms_max = std::numeric_limits<float>::min();
    float ms_min = std::numeric_limits<float>::max();
    float ms_total, ms_i;
    hipEvent_t start_i, stop_i;
    hipEventCreate(&start_i);
    hipEventCreate(&stop_i);
    ms_total = 0;

    hipProfilerStart();
    for (int i_=0; i_<steps; i_++)
    {
        hipEventRecord(start_i, 0);
        LSTM_DEV_FUNC
        hipEventRecord(stop_i, 0);
        hipEventSynchronize(stop_i);
        hipEventElapsedTime(&ms_i, start_i, stop_i);
        hipDeviceSynchronize();
        // printf("Iteration time %f ms\n", ms_i);
        ms_total += ms_i;
        if (ms_i > ms_max)  ms_max = ms_i;
        if (ms_i < ms_min) ms_min = ms_i;
    }
    hipProfilerStop();
    hipDeviceSynchronize();
    printf("Summary: [min, max, mean] = [%f, %f, %f] ms\n",  ms_min, ms_max, ms_total / steps);
    result = hipGetLastError();
    CUDA_CHECK_RESULT

    lstm_data.free();
}

int main(int argc, char** argv) {
    benchmark_lstm(argc, argv);
    return 0;
}
