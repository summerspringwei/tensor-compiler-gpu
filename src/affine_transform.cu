#include "hip/hip_runtime.h"
#include <iostream>
#include <list>
#include <vector>
#include <chrono>
#include <cstring>
#include <map>
#include <assert.h>

#include <hip/hip_runtime.h>

#include "affine_transform.hpp"

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


const int block_size = 256;
__global__ void affine_transform_kernel(float* target_coords, float* coords, float* trans, int batch, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= batch * n){
        return;
    }
    float x = coords[2 * idx];
    float y = coords[2 * idx + 1];
    target_coords[2 * idx] = trans[0] * x + trans[1] * y + trans[2] * 1;
    target_coords[2 * idx + 1] = trans[3] * x + trans[4] * y + trans[5] * 1;
}


/**
 * @brief 
 * 
 * @param coords shape (n, 2)
 * @param center shape (2)
 * @param scale shape(2)
 * @param output_size shape(2)
 */
void affine_transform(float* target_coords, float* coords, float* trans, int batch, int n, int loop_count){
    const int k = 2;
    // coords shape (b, n, 2)
    float* d_coords = NULL;
    hipError_t err = hipSuccess;
    err = hipMalloc((void **)&d_coords, sizeof(float)*n*k*batch);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector d_coords (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // trans shape (2, 3), save in opencv
    float *d_trans = NULL;
    err = hipMalloc((void **)&d_trans, sizeof(float) * 2 * 3);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector d_trans (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // target_cords shape (b, 100, 3), save in opencv
    float *d_target_coords = NULL;
    err = hipMalloc((void **)&d_target_coords, sizeof(float)*n*k*batch);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to allocate device vector d_target_coords (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_coords, coords, sizeof(float)*n*k, hipMemcpyHostToDevice);
    err = hipMemcpy(d_trans, trans, sizeof(float)*2*3, hipMemcpyHostToDevice);

    hipEvent_t startEvent, stopEvent;
    float ms = 0.0;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    
    auto t1 = std::chrono::steady_clock::now();
    dim3 threadsPerBlock(block_size);
    dim3 numBlocks(batch * n / threadsPerBlock.x + 1);
    // Launch the arnold CUDA Kernel
    checkCuda( hipEventRecord(startEvent,0));
    for(int i=0; i<loop_count; ++i){
        affine_transform_kernel<<<numBlocks, threadsPerBlock>>>(d_target_coords, d_coords, d_trans, batch, n);
        checkCuda( hipEventRecord(stopEvent,0));
    }
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    hipDeviceSynchronize();

    auto t2 = std::chrono::steady_clock::now();
    double latency = std::chrono::duration<double, std::micro>(t2-t1).count();
    printf("cudaEvent latency avg %f\n", ms / loop_count);
    // printf("[%f, %f] bandwidth %f\n", latency, ms,  n*sizeof(float) * 1e3 /1024/1024/ms);
    err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(target_coords, d_target_coords, sizeof(float) *n*k*batch, hipMemcpyDeviceToHost);

    hipFree(d_coords);
    hipFree(d_trans);
    hipFree(d_target_coords);
}

