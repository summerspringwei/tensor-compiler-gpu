#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <torch/extension.h>

inline hipError_t checkCuda(hipError_t result, char* msg)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "In %s CUDA Runtime Error: %s\n", msg, hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "In %s CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__ void print_cuda_kernel(float* input, long n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx != 0){
        return;
    }
    for(long i=0; i<n; ++i){
        printf("%f ", input[i]);
    }printf("\n");
}

void print_cuda(float* input, long n){
    printf("Start print_cuda\n");
    int block_size = 32;
    dim3 threadsPerBlock(block_size);
    dim3 numBlocks(1);
    print_cuda_kernel<<<numBlocks, threadsPerBlock>>>(input, n);
    checkCuda(hipDeviceSynchronize(), "print_cuda");
}

__global__ void print_cuda_kernel(torch::PackedTensorAccessor64<float, 1> input, long n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx != 0){
        return;
    }
    for(long i=0; i<n; ++i){
        printf("%f ", input[i]);
    }printf("\n");
}

void print_cuda(torch::PackedTensorAccessor64<float, 1> input, long n){
    printf("Start print_cuda\n");
    int block_size = 32;
    dim3 threadsPerBlock(block_size);
    dim3 numBlocks(1);
    print_cuda_kernel<<<numBlocks, threadsPerBlock>>>(input, n);
    checkCuda(hipDeviceSynchronize(), "print_cuda");
}
