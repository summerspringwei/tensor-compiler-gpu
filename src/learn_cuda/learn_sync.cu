#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>


__global__ void learn_grid_sync(float* input, float* output, const int totalNumber, const int blockSize){
  cooperative_groups::grid_group grid = cooperative_groups::this_grid();
  input[blockIdx.x * blockSize + threadIdx.x] += 1;
  grid.sync();
  output[blockIdx.x * blockSize + threadIdx.x] = input[totalNumber - 1 - (blockIdx.x * blockSize + threadIdx.x)];
}


int main(){
  const int totalNumber = 128 * 32;
  const int blockSize = 128;
  float* input = new float[totalNumber];
  float* output = new float[totalNumber];

  hipError_t err = hipSuccess;
  float* d_input = NULL, *d_output=NULL;
  hipMalloc((void **)&d_input, sizeof(float)*totalNumber);
  hipMalloc((void **)&d_output, sizeof(float)*totalNumber);
  for(int i=0; i<totalNumber; ++i){
    input[i] = i;
  }
  
  hipMemcpy(d_input, input, sizeof(float)*totalNumber, hipMemcpyHostToDevice);
  void * kernel_args[] = {
    (void*)&(d_input), (void*)&(d_output), (void*)&(totalNumber),(void*)&(blockSize)
  };

  hipLaunchCooperativeKernel((void*)learn_grid_sync, dim3(32, 1,1), dim3(128,1,1), kernel_args, 1024);
  hipDeviceSynchronize();
  hipMemcpy(output, d_output, sizeof(float)*totalNumber, hipMemcpyDeviceToHost);
  for(int i=0; i<totalNumber; ++i){
    printf("%f ", output[i]);
  }
  return 0;
}
