#include <iostream>
#include <list>
#include <vector>
#include <chrono>
#include <cstring>
#include <map>
#include <assert.h>

#include <hip/hip_runtime.h>


// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}


template<typename T, typename T2>
__global__ void sobel1x3_row_v1(T2* dst, T* src, T2* kernel, const uint64_t b, const uint64_t c, const uint64_t h, const uint64_t w){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx > h * w){
        return;
    }
    int col = idx % w;
    int row = idx / w;
    const uint64_t img_size = c * h * w;
    const uint64_t plane_size = h * w;
    
    for(uint64_t i=0; i<b; ++i) {
        for(uint64_t j=0; j<c; ++j){
            int total_offset = i * img_size + j * plane_size + row * w + col;
            // For mirror pad
            if(col==0){
                dst[total_offset + idx] = src[total_offset + 1] + (src[total_offset] << 1) + src[total_offset + 1];
            }else if(col==w-1){
                dst[total_offset + idx] = src[total_offset - 1] + (src[total_offset] << 1) + src[total_offset - 1];
            }else{
                dst[total_offset + idx] = src[total_offset - 1] + (src[total_offset] << 1) + src[total_offset + 1];
            }
        }
    }
}

// 先做内部的，再做边缘的
// 每个block做image的一行
template<typename T, typename T2>
__global__ void sobel1x3_row_v2(T2* dst, T* src, T2* kernel, const uint64_t b, const uint64_t c, const uint64_t h, const uint64_t w){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row >= h || col >= w){
        return;
    }
    
    const uint64_t img_size = c * h * w;
    const uint64_t plane_size = h * w;
    __shared__ T buf[4096];//
    for(uint64_t i=0; i<b; ++i) {
        for(uint64_t j=0; j<c; ++j){
            uint64_t total_offset = i * img_size + j * plane_size + row * w + col;
            if(row==0 && col > 0 && col < w-1){
                printf("%lu\n", total_offset);
            }
            buf[col] = src[total_offset];
            
            __syncthreads();
            if(col == 0){
                dst[total_offset] = src[total_offset + 1] + (src[total_offset] << 1) + src[total_offset + 1];
            }else if(col == w-1){
                dst[total_offset] = src[total_offset - 1] + (src[total_offset] << 1) + src[total_offset - 1];
            }else{
                dst[total_offset] = buf[col - 1] + (buf[col] << 1) + buf[col + 1];
                if(row==0 && col > 0 && col < w-1){
                    printf("%d (%d, %d, %d) %d %f\n", col, buf[col-1], buf[col], buf[col+1], total_offset, dst[total_offset]);
                }
            }
        }
    }
}

// Reduce the blockDim to reduce the number of blocks
template<typename T, typename T2>
__global__ void sobel1x3_row_v3(T2* dst, T* src, T2* kernel, const uint64_t b, const uint64_t c, const uint64_t h, const uint64_t w){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if(row >= h || col >= w){
        return;
    }
    
    const uint64_t img_size = c * h * w;
    const uint64_t plane_size = h * w;
    const int tile_size = 4;
    // const int stride = w / tile_size;

    __shared__ T buf[4096];//
    for(uint64_t i=0; i<b; ++i) {
        for(uint64_t j=0; j<c; ++j){
            int total_offset = i * img_size + j * plane_size + row * w + col * tile_size;
            buf[col * tile_size] = src[total_offset + 0];
            buf[col * tile_size + 1] = src[total_offset + 1];
            buf[col * tile_size + 2] = src[total_offset + 2];
            buf[col * tile_size + 3] = src[total_offset + 3];
            __syncthreads();
            if(col == 0){
                dst[total_offset] = src[total_offset + 1] + (src[total_offset] << 1) + src[total_offset + 1];
            }else if(col*tile_size == w-1){
                dst[total_offset] = src[total_offset - 1] + (src[total_offset] << 1) + src[total_offset - 1];
            }else{
                dst[total_offset] = buf[col - 1] + (buf[col] << 1) + buf[col + 1];
                dst[total_offset] = buf[col] + (buf[col + 1] << 1) + buf[col + 2];
                dst[total_offset] = buf[col + 1] + (buf[col + 2] << 1) + buf[col + 3];
                dst[total_offset] = buf[col + 2] + (buf[col + 3] << 1) + buf[col + 4];
            }
        }
    }
}


template<typename T, typename T2>
__global__ void sobel3x1_col_v1(T2* dst, T* src, T2* kernel, const uint64_t b, const uint64_t c, const uint64_t h, const uint64_t w){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx > h * w){
        return;
    }
    int col = idx % w;
    int row = idx / w;
    // const uint32_t tile_size = 256;
    const uint64_t img_size = c * h * w;
    const uint64_t plane_size = h * w;
    
    for(uint64_t i=0; i<b; ++i) {
        for(uint64_t j=0; j<c; ++j){
            int total_offset = i * img_size + j * plane_size;
            // For mirror pad
            if(row==0){
                dst[total_offset + idx] = 0;
            }else if(row==h-1){
                dst[total_offset + idx] = 0;
            }else{
                // dst[total_offset + idx] = -src[total_offset + (row-1) * w + col] + src[total_offset + (row+1) * w + col];
                dst[total_offset + idx] = -src[total_offset + idx - w] + src[total_offset + idx + w];
            }
        }
    }
}

template<typename T, typename T2>
__global__ void sobel3x1_col_v2(T2* dst, T* src, T2* kernel, const uint64_t b, const uint64_t c, const uint64_t h, const uint64_t w){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    // const uint32_t tile_size = 256;
    const uint64_t img_size = c * h * w;
    const uint64_t plane_size = h * w;
    // Put several rows to a tile
    const int tile_size = 16;
    __shared__ T buf[tile_size + 2][128];// blockDim.x equal to 128

    if((row+1) * tile_size > h || col > w){
        return;
    }

    for(uint64_t i=0; i<b; ++i) {
        for(uint64_t j=0; j<c; ++j){
            int total_offset = i * img_size + j * plane_size;
            // For mirror pad
            if(row==0){
                dst[total_offset + row * tile_size * w + col] = 0;
            }else if(row==h-1){
                dst[total_offset + row * tile_size * w + col] = 0;
            }else{
                // Pre load (tile_size + 2) rows into shared memory
                for(int k=0; k<tile_size+2; ++k){
                    buf[k][threadIdx.y] = src[total_offset + (row * tile_size - 1 + k) * w + col];
                }
                for(int k=2; k<tile_size+2; ++k){
                    dst[total_offset + (row * tile_size + (k-2)) * w + col] = -buf[k-2][threadIdx.y] + buf[k][threadIdx.y];
                }
            }
        }
    }
}

template<typename T, typename T2>
__global__ void sobel1x3(T2* dst, T* src, T2* kernel, const uint64_t b, const uint64_t c, const uint64_t h, const uint64_t w){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    // const uint32_t tile_size = 256;
    const uint64_t img_size = c * h * w;
    const uint64_t plane_size = h * w;
    // Put several rows to a tile
    const int tile_size = 16;
    const int block_size = 128;
    __shared__ T buf[tile_size + 2][block_size];// blockDim.x equal to 128

    if((row+1) * tile_size > h || col > w){
        return;
    }

    for(uint64_t i=0; i<b; ++i) {
        for(uint64_t j=0; j<c; ++j){
            int total_offset = i * img_size + j * plane_size;
            for(int k=0; k<tile_size; ++k){
                if(col == 0 || col == (w-1)){
                    continue;
                }
                int idx =  total_offset + (row * tile_size + k) * w + col;
                dst[idx] = src[idx - 1] + (src[idx] << 1) + src[idx + 1];
            }

            // For mirror pad
            if(row==0){
                dst[total_offset + row * tile_size * w + col] = 0;
            }else if(row==h-1){
                dst[total_offset + row * tile_size * w + col] = 0;
            }else{
                // Pre load (tile_size + 2) rows into shared memory
                for(int k=0; k<tile_size+2; ++k){
                    buf[k][threadIdx.y] = src[total_offset + (row * tile_size - 1 + k) * w + col];
                }
                for(int k=2; k<tile_size+2; ++k){
                    dst[total_offset + (row * tile_size + (k-2)) * w + col] = -buf[k-2][threadIdx.y] + buf[k][threadIdx.y];
                }
            }
        }
    }
}



uint64_t get_shape_size(std::vector<uint64_t> shape){
    uint64_t shape_size = 1;
    for(auto s: shape){
        shape_size *= s;
    }
    return shape_size;
}

int block_size = 128;

template<typename T, typename T2>
void sobel_cuda(T2* dst, T* src, T2* kernel, std::vector<uint64_t> shape, int loop_count){
    // In NCHW format
    // Error code to check return values for CUDA calls
    
    uint64_t n = shape[0], c = shape[1], h = shape[2], w = shape[3];
    T2* d_dst = NULL;
    hipError_t err = hipSuccess;
    const uint64_t elements_num = get_shape_size(shape);
    err = hipMalloc((void **)&d_dst, elements_num * sizeof(T2));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_a (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    T *d_src = NULL;
    err = hipMalloc((void **)&d_src, elements_num * sizeof(T));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(d_src, src, elements_num * sizeof(T), hipMemcpyHostToDevice);
    T2* d_kernel = NULL;
    err = hipMalloc((void **)&d_kernel, c*sizeof(T2));
    err = hipMemcpy(d_kernel, kernel, c * sizeof(T2), hipMemcpyHostToDevice);

    // cudaThreadSetCacheConfig(cudaFuncCachePreferL1);
    hipEvent_t startEvent, stopEvent;
    float total_time = 0.0;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );

    dim3 threadsPerBlockK1(1, w);
    dim3 numBlocksK1(h, 1);
    // Warm up
    checkCuda(hipEventRecord(startEvent,0) );
    sobel1x3_row_v2<T, T2><<<numBlocksK1, threadsPerBlockK1>>>(d_dst, d_src, d_kernel, n, c, h, w);
    dim3 threadsPerBlockK2(1, block_size);
    dim3 numBlocksK2(h/16, w/block_size);
    // sobel3x1_col_v2<T, T2><<<numBlocksK2, threadsPerBlockK2>>>(d_dst, d_src, d_kernel, n, c, h, w);
    checkCuda(hipEventRecord(stopEvent, 0) );
    checkCuda(hipEventSynchronize(stopEvent) );

    // CPU record latency
    auto t1 = std::chrono::steady_clock::now();
    for(int i=0; i<loop_count; ++i) {
        // sobel1x3_row_v1<T, T2><<<numBlocks, threadsPerBlock>>>(d_dst, d_src, d_kernel, n, c, h, w);
        sobel1x3_row_v2<T, T2><<<numBlocksK1, threadsPerBlockK1>>>(d_dst, d_src, d_kernel, n, c, h, w);
        // sobel3x1_col_v1<T, T2><<<numBlocks, threadsPerBlock>>>(d_dst, d_src, d_kernel, n, c, h, w);
        // sobel3x1_col_v2<T, T2><<<numBlocksK2, threadsPerBlockK2>>>(d_dst, d_src, d_kernel, n, c, h, w);
    }
    hipDeviceSynchronize();
    auto t2 = std::chrono::steady_clock::now();
    double latency = std::chrono::duration<double, std::micro>(t2-t1).count();
    
    // GPU record latency
    for(int i=0; i<loop_count; ++i) {
        float ms = 0.0;
        checkCuda( hipEventRecord(startEvent,0) );
        // sobel1x3_row_v1<T, T2><<<numBlocks, threadsPerBlock>>>(d_dst, d_src, d_kernel, n, c, h, w);
        sobel1x3_row_v2<T, T2><<<numBlocksK1, threadsPerBlockK1>>>(d_dst, d_src, d_kernel, n, c, h, w);
        // sobel3x1_col_v1<T, T2><<<numBlocks, threadsPerBlock>>>(d_dst, d_src, d_kernel, n, c, h, w);
        // sobel3x1_col_v2<T, T2><<<numBlocksK2, threadsPerBlockK2>>>(d_dst, d_src, d_kernel, n, c, h, w);
        checkCuda( hipEventRecord(stopEvent,0) );
        checkCuda( hipEventSynchronize(stopEvent) );
        checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
        total_time += ms;
    }
    
    printf("[%f, %f] bandwidth %f\n", latency / loop_count, total_time / loop_count,  elements_num*(sizeof(float) + sizeof(uint32_t)) * 1e3 /1024/1024/ (total_time/loop_count));
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    // printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(dst, d_dst, elements_num * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dst);
    hipFree(d_kernel);
}


int main(int argc, char** argv)
{
    if(argc < 2){
        printf("Usage: normalize img_size loop_count\n");
        return 0;
    }
    
    uint64_t n = atoi(argv[1]);
    int loop_count = atoi(argv[2]);
    assert((loop_count>0) && (n>0));
    const uint32_t img_size = 3*n*n;
    uint32_t* src = (uint32_t*)malloc(sizeof(uint32_t) * img_size);
    float* dst = (float*)malloc(sizeof(float) * img_size);
    float* kernel = (float*)malloc(sizeof(float) * 3);

    for(uint32_t i=0; i<img_size; ++i){
        src[i] = i % n;
    }

    std::vector<uint64_t> shape = {1, 3, n, n};
    auto t1 = std::chrono::steady_clock::now();
    sobel_cuda<uint32_t, float>(dst, src, kernel, shape, loop_count);

    printf("dst:\n");
    auto num_out = n*n*3;
    for(int i=0; i<(num_out < 16 ? num_out: 16);++i){
        printf("%f ", dst[i]);
    }printf("\n");
    auto t2 = std::chrono::steady_clock::now();
    printf("%ld %d %f\n", n, loop_count, std::chrono::duration<double, std::micro>(t2-t1).count() / loop_count);
    
    free(src);
    free(dst);
    free(kernel);
    return 0;
}

